#include "hip/hip_runtime.h"
/*
 *    GPU-accelerated scenario-based stochastic MPC for the operational
 *    management of drinking water networks.
 *    Copyright (C) 2017 Ajay. K. Sampathirao and P. Sopasakis
 *
 *    This library is free software; you can redistribute it and/or
 *    modify it under the terms of the GNU Lesser General Public
 *    License as published by the Free Software Foundation; either
 *    version 2.1 of the License, or (at your option) any later version.
 *
 *    This library is distributed in the hope that it will be useful,
 *    but WITHOUT ANY WARRANTY; without even the implied warranty of
 *    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 *    Lesser General Public License for more details.
 *
 *    You should have received a copy of the GNU Lesser General Public
 *    License along with this library; if not, write to the Free Software
 *    Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA 02110-1301 USA
 */
#include <cuda_device_runtime_api.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "rapidjson/document.h"
#include "rapidjson/rapidjson.h"
#include "rapidjson/filereadstream.h"
#include "rapidjson/filewritestream.h"
#include "rapidjson/prettywriter.h"
#include "SmpcController.cuh"



SmpcController::SmpcController(Forecaster *myForecaster, Engine *myEngine, SmpcConfiguration *mySmpcConfig){
	ptrMyForecaster = myForecaster;
	ptrMyEngine = myEngine;
	ptrMySmpcConfig = mySmpcConfig;
	DwnNetwork* ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree* ptrMyScenarioTree = ptrMyEngine->getScenarioTree();

	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t ns = ptrMyScenarioTree->getNumScenarios();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	stepSize = ptrMySmpcConfig->getStepSize();
	factorStepFlag = false;
	simulatorFlag = true;

	_CUDA( hipMalloc((void**)&devVecX, nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecU, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecV, nv*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecAcceleratedXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecAcceleratedPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPrimalXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPrimalPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDualXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDualPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecUpdateXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecUpdatePsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPrimalInfeasibilty, (2*nx + nu)*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecQ, ns*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecR, ns*nv*sizeof(real_t)) );

	_CUDA( hipMalloc((void**)&devPtrVecX, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecU, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecV, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecAcceleratedXi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecAcceleratedPsi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecPrimalXi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecPrimalPsi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecQ, ns*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecR, ns*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devControlAction, nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devStateUpdate, nx*sizeof(real_t)) );

	real_t** ptrVecX = new real_t*[nodes];
	real_t** ptrVecU = new real_t*[nodes];
	real_t** ptrVecV = new real_t*[nodes];
	real_t** ptrVecAcceleratedXi = new real_t*[nodes];
	real_t** ptrVecAcceleratedPsi = new real_t*[nodes];
	real_t** ptrVecPrimalXi = new real_t*[nodes];
	real_t** ptrVecPrimalPsi = new real_t*[nodes];
	real_t** ptrVecQ = new real_t*[ns];
	real_t** ptrVecR = new real_t*[ns];

	for(uint_t iScenario = 0; iScenario < ns; iScenario++){
		ptrVecQ[iScenario] = &devVecQ[iScenario*nx];
		ptrVecR[iScenario] = &devVecR[iScenario*nv];
	}
	for(uint_t iNode = 0; iNode < nodes; iNode++){
		ptrVecX[iNode] = &devVecX[iNode*nx];
		ptrVecU[iNode] = &devVecU[iNode*nu];
		ptrVecV[iNode] = &devVecV[iNode*nv];
		ptrVecAcceleratedXi[iNode] = &devVecAcceleratedXi[2*iNode*nx];
		ptrVecAcceleratedPsi[iNode] = &devVecAcceleratedPsi[iNode*nu];
		ptrVecPrimalXi[iNode] = &devVecPrimalXi[2*iNode*nx];
		ptrVecPrimalPsi[iNode] = &devVecPrimalPsi[iNode*nu];
	}

	_CUDA( hipMemset(devVecU, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecAcceleratedXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecAcceleratedPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecUpdateXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecUpdatePsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecDualXi, 0, 2*nx*nodes*sizeof(real_t)));
	_CUDA( hipMemset(devVecDualPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalInfeasibilty, 0, (2*nx + nu)*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecR, 0, ns*nv*sizeof(real_t)) );
	_CUDA( hipMemset(devVecQ, 0, ns*nx*sizeof(real_t)) );
	_CUDA( hipMemset(devControlAction, 0, nu*sizeof(real_t)) );
	_CUDA( hipMemset(devStateUpdate, 0, nx*sizeof(real_t)) );

	_CUDA( hipMemcpy(devPtrVecX, ptrVecX, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecU, ptrVecU, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecV, ptrVecV, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecAcceleratedXi, ptrVecAcceleratedXi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecAcceleratedPsi, ptrVecAcceleratedPsi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecPrimalXi, ptrVecPrimalXi, nodes*sizeof(real_t*),hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecPrimalPsi, ptrVecPrimalPsi, nodes*sizeof(real_t*),hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecQ, ptrVecQ, ns*sizeof(real_t*), hipMemcpyHostToDevice));
	_CUDA( hipMemcpy(devPtrVecR, ptrVecR, ns*sizeof(real_t*), hipMemcpyHostToDevice));

	vecPrimalInfs = new real_t[ptrMySmpcConfig->getMaxIterations()];

	economicKpi = 0;
	smoothKpi = 0;
	safeKpi = 0;
	networkKpi = 0;

	delete [] ptrVecX;
	delete [] ptrVecU;
	delete [] ptrVecV;
	delete [] ptrVecAcceleratedXi;
	delete [] ptrVecAcceleratedPsi;
	delete [] ptrVecPrimalXi;
	delete [] ptrVecPrimalPsi;
	delete [] ptrVecQ;
	delete [] ptrVecR;
	ptrVecX = NULL;
	ptrVecU = NULL;
	ptrVecV = NULL;
	ptrVecAcceleratedXi = NULL;
	ptrVecAcceleratedPsi = NULL;
	ptrVecPrimalXi = NULL;
	ptrVecPrimalPsi = NULL;
	ptrVecQ = NULL;
	ptrVecR = NULL;
	ptrMyNetwork = NULL;
	ptrMyScenarioTree = NULL;
}

/**
 * Construct a new Controller with a given engine.
 * @param  pathToConfigFile   path to the controller configuration file
 */
SmpcController::SmpcController(string pathToConfigFile){
	ptrMySmpcConfig = new SmpcConfiguration( pathToConfigFile );
	string pathToForecaster = ptrMySmpcConfig->getPathToForecaster();
	ptrMyForecaster = new Forecaster( pathToForecaster );
	ptrMyEngine = new Engine( ptrMySmpcConfig );

	stepSize = ptrMySmpcConfig->getStepSize();
	factorStepFlag = false;
	simulatorFlag = true;
	bool globalFbeStatus = ptrMyEngine->getGlobalFbeFlag();

	vecPrimalInfs = new real_t[ptrMySmpcConfig->getMaxIterations()];

	economicKpi = 0;
	smoothKpi = 0;
	safeKpi = 0;
	networkKpi = 0;

	if( globalFbeStatus )

	else
		this->allocateApgAlgorithm();

	//ptrMyNetwork = NULL;
	//ptrMyScenarioTree = NULL;
}

/*
 * Allocate memory of the APG algorithm
 *   - dual and accelerated vectors (psi, xi)
 *   - primal variables (x, u and t)
 *   - primal infeasibility (Hz - t)
 */
void SmpcController::allocateApgAlgorithm(){
	DwnNetwork* ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree* ptrMyScenarioTree = ptrMyEngine->getScenarioTree();

	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t ns = ptrMyScenarioTree->getNumScenarios();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();

	_CUDA( hipMalloc((void**)&devVecX, nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecU, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecV, nv*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecAcceleratedXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecAcceleratedPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPrimalXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPrimalPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDualXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDualPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecUpdateXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecUpdatePsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPrimalInfeasibilty, (2*nx + nu)*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecQ, ns*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecR, ns*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devControlAction, nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devStateUpdate, nx*sizeof(real_t)) );

	_CUDA( hipMalloc((void**)&devPtrVecX, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecU, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecV, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecAcceleratedXi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecAcceleratedPsi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecPrimalXi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecPrimalPsi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecQ, ns*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecR, ns*sizeof(real_t*)) );

	real_t** ptrVecX = new real_t*[nodes];
	real_t** ptrVecU = new real_t*[nodes];
	real_t** ptrVecV = new real_t*[nodes];
	real_t** ptrVecAcceleratedXi = new real_t*[nodes];
	real_t** ptrVecAcceleratedPsi = new real_t*[nodes];
	real_t** ptrVecPrimalXi = new real_t*[nodes];
	real_t** ptrVecPrimalPsi = new real_t*[nodes];
	real_t** ptrVecQ = new real_t*[ns];
	real_t** ptrVecR = new real_t*[ns];

	for(uint_t iScenario = 0; iScenario < ns; iScenario++){
		ptrVecQ[iScenario] = &devVecQ[iScenario*nx];
		ptrVecR[iScenario] = &devVecR[iScenario*nv];
	}
	for(uint_t iNode = 0; iNode < nodes; iNode++){
		ptrVecX[iNode] = &devVecX[iNode*nx];
		ptrVecU[iNode] = &devVecU[iNode*nu];
		ptrVecV[iNode] = &devVecV[iNode*nv];
		ptrVecAcceleratedXi[iNode] = &devVecAcceleratedXi[2*iNode*nx];
		ptrVecAcceleratedPsi[iNode] = &devVecAcceleratedPsi[iNode*nu];
		ptrVecPrimalXi[iNode] = &devVecPrimalXi[2*iNode*nx];
		ptrVecPrimalPsi[iNode] = &devVecPrimalPsi[iNode*nu];
	}

	_CUDA( hipMemset(devVecU, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecAcceleratedXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecAcceleratedPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecUpdateXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecUpdatePsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecDualXi, 0, 2*nx*nodes*sizeof(real_t)));
	_CUDA( hipMemset(devVecDualPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalInfeasibilty, 0, (2*nx + nu)*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecR, 0, ns*nv*sizeof(real_t)) );
	_CUDA( hipMemset(devVecQ, 0, ns*nx*sizeof(real_t)) );
	_CUDA( hipMemset(devControlAction, 0, nu*sizeof(real_t)) );
	_CUDA( hipMemset(devStateUpdate, 0, nx*sizeof(real_t)) );

	_CUDA( hipMemcpy(devPtrVecX, ptrVecX, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecU, ptrVecU, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecV, ptrVecV, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecAcceleratedXi, ptrVecAcceleratedXi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecAcceleratedPsi, ptrVecAcceleratedPsi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecPrimalXi, ptrVecPrimalXi, nodes*sizeof(real_t*),hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecPrimalPsi, ptrVecPrimalPsi, nodes*sizeof(real_t*),hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecQ, ptrVecQ, ns*sizeof(real_t*), hipMemcpyHostToDevice));
	_CUDA( hipMemcpy(devPtrVecR, ptrVecR, ns*sizeof(real_t*), hipMemcpyHostToDevice));

	delete [] ptrVecX;
	delete [] ptrVecU;
	delete [] ptrVecV;
	delete [] ptrVecAcceleratedXi;
	delete [] ptrVecAcceleratedPsi;
	delete [] ptrVecPrimalXi;
	delete [] ptrVecPrimalPsi;
	delete [] ptrVecQ;
	delete [] ptrVecR;
	ptrVecX = NULL;
	ptrVecU = NULL;
	ptrVecV = NULL;
	ptrVecAcceleratedXi = NULL;
	ptrVecAcceleratedPsi = NULL;
	ptrVecPrimalXi = NULL;
	ptrVecPrimalPsi = NULL;
	ptrVecQ = NULL;
	ptrVecR = NULL;

}

/**
 * Performs the initialise the smpc controller
 *   - update the current state and previous controls in the device memory
 *   - perform the factor step
 */
void SmpcController::initialiseSmpcController(){
	factorStepFlag = true;
	real_t *currentX = ptrMySmpcConfig->getCurrentX();
	real_t *prevU = ptrMySmpcConfig->getPrevU();
	real_t *prevDemand = ptrMySmpcConfig->getPrevDemand();

	ptrMyEngine->factorStep();
	ptrMyEngine->updateStateControl(currentX, prevU, prevDemand);
	ptrMyEngine->eliminateInputDistubanceCoupling( ptrMyForecaster->getNominalDemand(),
			ptrMyForecaster->getNominalPrices());
	/*
	uint_t nx = ptrMySmpcConfig->getNX();
	uint_t nu = ptrMySmpcConfig->getNU();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t nodes = this->getScenarioTree()->getNumNodes();
	uint_t N = ptrMyForecaster->getPredHorizon();

	currentX = NULL;
	prevU = NULL;
	prevDemand = NULL;
	*/
}

/**
 * Performs the dual extrapolation step with given parameter.
 * @param extrapolation parameter.
 */
void SmpcController::dualExtrapolationStep(real_t lambda){
	DwnNetwork* ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree* ptrMyScenarioTree = ptrMyEngine->getScenarioTree();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	real_t alpha;
	// w = (1 + \lambda)y_k - \lambda y_{k-1}
	_CUDA(hipMemcpy(devVecAcceleratedXi, devVecUpdateXi, 2*nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice) );
	_CUDA(hipMemcpy(devVecAcceleratedPsi, devVecUpdatePsi, nu*nodes*sizeof(real_t), hipMemcpyDeviceToDevice) );
	alpha = 1 + lambda;
	_CUBLAS(hipblasSscal(ptrMyEngine->getCublasHandle(), 2*nx*nodes, &alpha, devVecAcceleratedXi, 1));
	_CUBLAS(hipblasSscal(ptrMyEngine->getCublasHandle(), nu*nodes, &alpha, devVecAcceleratedPsi, 1));
	alpha = -lambda;
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), 2*nx*nodes, &alpha, devVecXi, 1, devVecAcceleratedXi, 1));
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu*nodes, &alpha, devVecPsi, 1, devVecAcceleratedPsi, 1));
	// y_{k} = y_{k-1}
	_CUDA(hipMemcpy(devVecXi, devVecUpdateXi, 2*nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	_CUDA(hipMemcpy(devVecPsi, devVecUpdatePsi, nu*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	//ptrMyNetwork = NULL;
	//ptrMyScenarioTree = NULL;
}

/**
 * Computes the dual gradient.This is the main computational
 * algorithm for the proximal gradient algorithm
 */
void SmpcController::solveStep(){
	DwnNetwork *ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree *ptrMyScenarioTree = ptrMyEngine->getScenarioTree();
	real_t *devTempVecR, *devTempVecQ, *devLv;
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t ns = ptrMyScenarioTree->getNumScenarios();
	uint_t N =  ptrMyScenarioTree->getPredHorizon();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t *nodesPerStage = ptrMyScenarioTree->getNodesPerStage();
	uint_t *nodesPerStageCumul = ptrMyScenarioTree->getNodesPerStageCumul();
	uint_t iStageCumulNodes, iStageNodes, prevStageNodes, prevStageCumulNodes;
	real_t scale[3] = {-0.5, 1, -1};
	real_t alpha = 1;
	real_t beta = 0;
	real_t *xHost = new real_t[nu*nodes];

	if(factorStepFlag == false){
		initialiseSmpcController();
		factorStepFlag = true;
	}

	_CUDA( hipMalloc((void**)&devTempVecQ, ns*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devTempVecR, ns*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devLv, ns*nu*sizeof(real_t)) );
	_CUDA( hipMemcpy(ptrMyEngine->getMatSigma(), ptrMyEngine->getVecBeta(), nv*nodes*sizeof(real_t),
			hipMemcpyDeviceToDevice) );

	real_t *x = new real_t[ns*nv*nv];

	//Backward substitution
	for(uint_t iStage = N-1;iStage > -1;iStage--){
		iStageCumulNodes = nodesPerStageCumul[iStage];
		iStageNodes = nodesPerStage[iStage];

		if(iStage < N-1){
			// sigma=sigma+r
			_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), iStageNodes*nv, &alpha, devVecR, 1,
					&ptrMyEngine->getMatSigma()[iStageCumulNodes*nv],1));
		}

		// v=Omega*sigma
		_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nv,
				&scale[0], (const real_t**)&ptrMyEngine->getPtrMatOmega()[iStageCumulNodes], nv,
				(const real_t**)&ptrMyEngine->getPtrMatSigma()[iStageCumulNodes], nv, &beta,
				&devPtrVecV[iStageCumulNodes], nv, iStageNodes));

		if(iStage < N-1){
			// v=Theta*q+v
			_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nx,
					&alpha, (const real_t**)&ptrMyEngine->getPtrMatTheta()[iStageCumulNodes], nv,
					(const real_t**)devPtrVecQ, nx, &alpha, &devPtrVecV[iStageCumulNodes], nv, iStageNodes));
		}

		// v=Psi*psi+v
		_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nu, &alpha,
				(const real_t**)&ptrMyEngine->getPtrMatPsi()[iStageCumulNodes], nv,
				(const real_t**)&devPtrVecAcceleratedPsi[iStageCumulNodes], nu, &alpha, &devPtrVecV
				[iStageCumulNodes], nv, iStageNodes));

		// v=Phi*xi+v
		_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, 2*nx, &alpha,
				(const real_t**)&ptrMyEngine->getPtrMatPhi()[iStageCumulNodes], nv, (const real_t**)
				&devPtrVecAcceleratedXi[iStageCumulNodes], 2*nx, &alpha, &devPtrVecV[iStageCumulNodes],
				nv, iStageNodes));

		// r=sigma
		_CUDA(hipMemcpy(devVecR, &ptrMyEngine->getMatSigma()[iStageCumulNodes*nv], nv*iStageNodes*sizeof(real_t),
				hipMemcpyDeviceToDevice));

		// r=D*xi+r
		_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, 2*nx, &alpha,
				(const real_t**)&ptrMyEngine->getPtrMatD()[iStageCumulNodes], nv, (const real_t**)
				&devPtrVecAcceleratedXi[iStageCumulNodes], 2*nx, &alpha, devPtrVecR, nv, iStageNodes));

		// r=f*psi+r
		_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nu, &alpha,
				(const real_t**)&ptrMyEngine->getPtrMatF()[iStageCumulNodes], nv, (const real_t**)
				&devPtrVecAcceleratedPsi[iStageCumulNodes], nu, &alpha, devPtrVecR, nv, iStageNodes));

		if(iStage < N-1){
			// r=g*q+r
			//_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nx, &alpha,
			//		(const real_t**)&ptrMyEngine->getPtrMatG()[iStageCumulNodes], nv, (const real_t**)devPtrVecQ,
			//		nx, &alpha, devPtrVecR, nv, iStageNodes));
			_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nx, &alpha,
					(const real_t**)&ptrMyEngine->getPtrMatG()[0], nv, (const real_t**)devPtrVecQ,
					nx, &alpha, devPtrVecR, nv, iStageNodes));
		}

		if(iStage < N-1){
			// q=F'xi+q
			_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_T, HIPBLAS_OP_N, nx, 1, 2*nx, &alpha,
					(const real_t**)&ptrMyEngine->getPtrSysMatF()[iStageCumulNodes], 2*nx, (const real_t**)
					&devPtrVecAcceleratedXi[iStageCumulNodes], 2*nx, &alpha, devPtrVecQ, nx, iStageNodes));
		}else{
			// q=F'xi
			_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_T, HIPBLAS_OP_N, nx, 1, 2*nx, &alpha,
					(const real_t**)&ptrMyEngine->getPtrSysMatF()[iStageCumulNodes], 2*nx, (const real_t**)
					&devPtrVecAcceleratedXi[iStageCumulNodes], 2*nx, &beta, devPtrVecQ, nx, iStageNodes));
		}

		if(iStage > 0){
			prevStageNodes = nodesPerStage[iStage - 1];
			prevStageCumulNodes = nodesPerStageCumul[iStage - 1];
			if( (iStageNodes - prevStageNodes) > 0 ){
				solveSumChildren<<<prevStageNodes, nx>>>(devVecQ, devTempVecQ, ptrMyEngine->getTreeNumChildren(),
						ptrMyEngine->getTreeNumChildrenCumul(), prevStageCumulNodes, prevStageNodes, iStage - 1, nx);
				solveSumChildren<<<prevStageNodes, nv>>>(devVecR, devTempVecR, ptrMyEngine->getTreeNumChildren(),
						ptrMyEngine->getTreeNumChildrenCumul(), prevStageCumulNodes, prevStageNodes, iStage - 1 , nv);
				_CUDA(hipMemcpy(devVecR, devTempVecR, prevStageNodes*nv*sizeof(real_t), hipMemcpyDeviceToDevice));
				_CUDA(hipMemcpy(devVecQ, devTempVecQ, prevStageNodes*nx*sizeof(real_t), hipMemcpyDeviceToDevice));
			}
		}
	}

	// Forward substitution
	_CUDA(hipMemcpy(devVecU, ptrMyEngine->getVecUhat(), nodes*nu*sizeof(real_t), hipMemcpyDeviceToDevice));

	for(uint_t iStage = 0;iStage < N;iStage++){
		iStageNodes = nodesPerStage[iStage];
		iStageCumulNodes = nodesPerStageCumul[iStage];
		if(iStage == 0){
			// u = prevU - prevUhat
			_CUBLAS( hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu, &alpha, ptrMyEngine->getVecPreviousControl(), 1,
					devVecU, 1));
			_CUBLAS( hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu, &scale[2], ptrMyEngine->getVecPreviousUhat(), 1,
					devVecU, 1));
			// x=p
			_CUDA( hipMemcpy(devVecX, ptrMyEngine->getVecCurrentState(), nx*sizeof(real_t), hipMemcpyDeviceToDevice) );
			// x=x+w
			_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nx, &alpha, ptrMyEngine->getVecE(), 1, devVecX, 1));
			// u=Lv+\hat{u}
			_CUBLAS(hipblasSgemv(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, nu, nv, &alpha,
					ptrMyEngine->getSysMatL(), nu, devVecV, 1, &alpha, devVecU, 1) );
			// x=x+Bu
			_CUBLAS(hipblasSgemv(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, nx, nu, &alpha,
					ptrMyEngine->getSysMatB(), nx, devVecU, 1, &alpha, devVecX, 1) );
		}else{
			prevStageCumulNodes = nodesPerStageCumul[iStage - 1];
			if((nodesPerStage[iStage] - nodesPerStage[iStage-1]) > 0){
				// u_k=Lv_k+\hat{u}_k
				_CUBLAS(hipblasSgemm(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nu, iStageNodes, nv,
						&alpha, ptrMyEngine->getSysMatL(), nu, &devVecV[iStageCumulNodes*nv], nv, &alpha,
						&devVecU[iStageCumulNodes*nu], nu));
				// prevLv = u_{k-1} - uHat_{k-1}
				_CUDA( hipMemcpy(devLv, &devVecU[prevStageCumulNodes*nu], nu*nodesPerStage[iStage-1]*sizeof(real_t), hipMemcpyDeviceToDevice));
				_CUBLAS( hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu*nodesPerStage[iStage-1], &scale[2],
						&ptrMyEngine->getVecUhat()[prevStageCumulNodes*nu], 1, devLv, 1));
				// u_{k} = u_{k} + prevLu
				solveChildNodesUpdate<<<iStageNodes, nu>>>(devLv, &devVecU[iStageCumulNodes*nu], ptrMyEngine->getTreeAncestor(),
						iStageCumulNodes, nu);
				// x=w
				_CUDA(hipMemcpy(&devVecX[iStageCumulNodes*nx], &ptrMyEngine->getVecE()[iStageCumulNodes*nx],
						iStageNodes*nx*sizeof(real_t), hipMemcpyDeviceToDevice));
				// x=x+Bu
				_CUBLAS(hipblasSgemm(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nx, iStageNodes, nu, &alpha,
						ptrMyEngine->getSysMatB(), nx, &devVecU[iStageCumulNodes*nu], nu, &alpha, &devVecX[iStageCumulNodes*nx], nx));
				// x_{k+1}=x_k
				solveChildNodesUpdate<<<iStageNodes, nx>>>(&devVecX[prevStageCumulNodes*nx], &devVecX[iStageCumulNodes*nx],
						ptrMyEngine->getTreeAncestor(), iStageCumulNodes, nx);
			}else{
				// u_k = u_{k-1} - uHat_{k-1}
				_CUBLAS( hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu*iStageNodes, &alpha, &devVecU[prevStageCumulNodes*nu], 1,
						&devVecU[iStageCumulNodes*nu], 1));
				_CUBLAS( hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu*iStageNodes, &scale[2], &ptrMyEngine->getVecUhat()
						[prevStageCumulNodes*nu], 1, &devVecU[iStageCumulNodes*nu], 1));
				// u_k=Lv_k+\hat{u}_k + u_k
				_CUBLAS(hipblasSgemm(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nu, iStageNodes, nv, &alpha,
						ptrMyEngine->getSysMatL(), nu, &devVecV[iStageCumulNodes*nv], nv, &alpha, &devVecU[iStageCumulNodes*nu], nu));
				// x_{k+1}=x_{k}
				_CUDA(hipMemcpy(&devVecX[iStageCumulNodes*nx], &devVecX[prevStageCumulNodes*nx], nx*iStageNodes*sizeof(real_t),
						hipMemcpyDeviceToDevice));
				// x=x+w
				_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nx*iStageNodes, &alpha, &ptrMyEngine->getVecE()
						[iStageCumulNodes*nx], 1, &devVecX[iStageCumulNodes*nx], 1));
				// x=x+Bu
				_CUBLAS(hipblasSgemm(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nx, iStageNodes, nu, &alpha,
						ptrMyEngine->getSysMatB(), nx, &devVecU[iStageCumulNodes*nu], nu, &alpha, &devVecX[iStageCumulNodes*nx], nx));
			}

		}
	}

	_CUDA(hipFree(devTempVecQ));
	_CUDA(hipFree(devTempVecR));
	_CUDA(hipFree(devLv) );
	devTempVecQ = NULL;
	devTempVecR = NULL;
	devLv = NULL;
	//ptrMyNetwork = NULL;
	//ptrMyScenarioTree = NULL;
}

/**
 * Computes the proximal operator of g at the current point and updates
 * (primal psi, primal xi) - Hx, (dual psi, dual xi) - z.
 */
void SmpcController::proximalFunG(){
	DwnNetwork *ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree *ptrMyScenarioTree = ptrMyEngine->getScenarioTree();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	real_t alpha = 1;
	real_t negAlpha = -1;
	real_t beta = 0;
	real_t penaltyScalar;
	real_t invLambda = 1/stepSize;
	real_t distanceXs, distanceXcst;
	real_t *devSuffleVecXi;
	real_t *devVecDiffXi;
	_CUDA( hipMalloc((void**)&devSuffleVecXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDiffXi, 2*nx*nodes*sizeof(real_t)) );

	// primalX = Hx
	_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 2*nx, 1, nx, &alpha, (const real_t**)
			ptrMyEngine->getPtrSysMatF(), 2*nx, (const real_t**)devPtrVecX, nx, &beta, devPtrVecPrimalXi, 2*nx, nodes) );
	_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nu, 1, nu, &alpha, (const real_t**)
			ptrMyEngine->getPtrSysMatG(), nu, (const real_t**)devPtrVecU, nu, &beta, devPtrVecPrimalPsi, nu, nodes) );

	// Hx + \lambda^{-1}w
	_CUDA( hipMemcpy(devVecDualXi, devVecPrimalXi, 2*nodes*nx*sizeof(real_t), hipMemcpyDeviceToDevice) );
	_CUDA( hipMemcpy(devVecDualPsi, devVecPrimalPsi, nodes*nu*sizeof(real_t), hipMemcpyDeviceToDevice) );
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), 2*nx*nodes, &invLambda, devVecAcceleratedXi, 1, devVecDualXi, 1) );
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu*nodes, &invLambda, devVecAcceleratedPsi, 1, devVecDualPsi, 1) );

	_CUDA( hipMemcpy(devVecDiffXi, devVecDualXi, 2*nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice) );
	// proj(xi|X), proj(xi|Xsafe)
	projectionBox<<<nodes, nx>>>(devVecDualXi, ptrMyEngine->getSysXmin(), ptrMyEngine->getSysXmax(), 2*nx, 0, nx*nodes);
	projectionBox<<<nodes, nx>>>(devVecDualXi, ptrMyEngine->getSysXs(), ptrMyEngine->getSysXsUpper(), 2*nx, nx, nx*nodes);

	// x-proj(x)
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), 2*nx*nodes, &negAlpha, devVecDualXi, 1, devVecDiffXi, 1) );
	shuffleVector<<<nodes, 2*nx>>>(devSuffleVecXi, devVecDiffXi, nx, 2, nodes);
	//distance with constraints X
	_CUBLAS(hipblasSnrm2(ptrMyEngine->getCublasHandle(), nx*nodes, devSuffleVecXi, 1, &distanceXcst));
	if(distanceXcst > invLambda*ptrMySmpcConfig->getPenaltyState()){
		//cout << " prox distance ";
		penaltyScalar = 1 - invLambda*ptrMySmpcConfig->getPenaltyState()/distanceXcst;
		additionVectorOffset<<<nodes, nx>>>(devVecDualXi, devVecDiffXi, penaltyScalar, 2*nx, 0, nx*nodes);
	}
	//distance with Xsafe
	_CUBLAS(hipblasSnrm2(ptrMyEngine->getCublasHandle(), nx*nodes, &devSuffleVecXi[nx*nodes], 1, &distanceXs));
	if(distanceXs > invLambda*ptrMySmpcConfig->getPenaltySafety()){
		//cout << " prox distance ";
		penaltyScalar = 1-invLambda*ptrMySmpcConfig->getPenaltySafety()/distanceXs;
		additionVectorOffset<<<nodes, nx>>>(devVecDualXi, devVecDiffXi, penaltyScalar, 2*nx, nx, nx*nodes);
	}
	//cout << " distance is " << distanceXcst << " " << distanceXs << endl;
	/**/
	projectionBox<<<nodes, nu>>>(devVecDualPsi, ptrMyEngine->getSysUmin(), ptrMyEngine->getSysUmax(), nu, 0, nu*nodes);
	_CUDA( hipFree(devSuffleVecXi) );
	_CUDA( hipFree(devVecDiffXi) );
	devSuffleVecXi = NULL;
	devVecDiffXi = NULL;
	//ptrMyNetwork = NULL;
	//ptrMyScenarioTree = NULL;
}

/**
 * Performs the update of the dual vector.
 */
void SmpcController::dualUpdate(){
	DwnNetwork *ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree *ptrMyScenarioTree = ptrMyEngine->getScenarioTree();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	real_t negAlpha = -1;
	//Hx - z
	_CUDA(hipMemcpy(devVecPrimalInfeasibilty, devVecPrimalXi, 2*nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	_CUDA(hipMemcpy(&devVecPrimalInfeasibilty[2*nx*nodes], devVecPrimalPsi, nu*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), 2*nx*nodes, &negAlpha, devVecDualXi, 1, devVecPrimalInfeasibilty, 1));
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu*nodes, &negAlpha, devVecDualPsi, 1, &devVecPrimalInfeasibilty[2*nx*nodes], 1));
	// y = w + \lambda(Hx - z)
	_CUDA( hipMemcpy(devVecUpdateXi, devVecAcceleratedXi, 2*nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	_CUDA( hipMemcpy(devVecUpdatePsi, devVecAcceleratedPsi, nu*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), 2*nx*nodes, &stepSize, devVecPrimalInfeasibilty, 1, devVecUpdateXi, 1) );
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu*nodes, &stepSize, &devVecPrimalInfeasibilty[2*nx*nodes], 1,
			devVecUpdatePsi, 1) );
	ptrMyNetwork = NULL;
	ptrMyScenarioTree = NULL;
}

/**
 * This method executes the APG algorithm and returns the primal infeasibility.
 * @return primalInfeasibilty;
 */
uint_t SmpcController::algorithmApg(){
	DwnNetwork *ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree *ptrMyScenarioTree = ptrMyEngine->getScenarioTree();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	_CUDA( hipMemset(devVecXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecAcceleratedXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecAcceleratedPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecUpdateXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecUpdatePsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecDualXi, 0, 2*nx*nodes*sizeof(real_t)));
	_CUDA( hipMemset(devVecDualPsi, 0, nu*nodes*sizeof(real_t)) );

	//cout << "number of nodes " <<nodes << endl;
	real_t theta[2] = {1, 1};
	real_t lambda;
	uint_t maxIndex;
	for (uint_t iter = 0; iter < ptrMySmpcConfig->getMaxIterations(); iter++){
	//for (uint_t iter = 0; iter < 40; iter++){
		lambda = theta[1]*(1/theta[0] - 1);
		dualExtrapolationStep(lambda);
		solveStep();
		proximalFunG();
		dualUpdate();
		theta[0] = theta[1];
		theta[1] = 0.5*(sqrt(pow(theta[1], 4) + 4*pow(theta[1], 2)) - pow(theta[1], 2));
		_CUBLAS( hipblasIsamax(ptrMyEngine->getCublasHandle(), (2*nx + nu)*nodes, devVecPrimalInfeasibilty,
				1, &maxIndex));
		_CUDA( hipMemcpy(&vecPrimalInfs[iter], &devVecPrimalInfeasibilty[maxIndex - 1], sizeof(real_t),
				hipMemcpyDeviceToHost));
	}
	//cout << endl;

	return 1;
}

/**
 * Invoke the SMPC controller on the current state of the network.
 * This method invokes #updateStateControl, eliminateInputDistubanceCoupling
 * and finally #algorithmApg.
 */
void SmpcController::controllerSmpc(){
	ptrMyEngine->updateStateControl(ptrMySmpcConfig->getCurrentX(), ptrMySmpcConfig->getPrevU(),
			ptrMySmpcConfig->getPrevDemand());
	ptrMyEngine->eliminateInputDistubanceCoupling(ptrMyForecaster->getNominalDemand(),
			ptrMyForecaster->getNominalPrices());
	algorithmApg();
}

/**
 * Computes a control action and returns a status code
 * which is an integer (1 = success).
 * @param u pointer to computed control action (CPU variable)
 * @return status code
 */
uint_t SmpcController::controlAction(real_t* u){
	uint_t status;
	size_t initialFreeBytes;
	size_t totalBytes;
	size_t finalFreeBytes;
	_CUDA( hipMemGetInfo(&initialFreeBytes, &totalBytes) );
	ptrMyEngine->updateStateControl(ptrMySmpcConfig->getCurrentX(), ptrMySmpcConfig->getPrevU(),
			ptrMySmpcConfig->getPrevDemand());
	ptrMyEngine->eliminateInputDistubanceCoupling(ptrMyForecaster->getNominalDemand(),
			ptrMyForecaster->getNominalPrices());
	status = algorithmApg();
	_CUDA( hipMemcpy(u, devVecU, ptrMySmpcConfig->getNU()*sizeof(real_t), hipMemcpyDeviceToHost));
	_CUDA( hipMemGetInfo(&finalFreeBytes, &totalBytes) );
	if( abs(finalFreeBytes - initialFreeBytes) > 0 ){
		cout << "RUNTIME ERROR: MEMORY LEAKS" << endl;
		return 0;
	}else
		return status;
}

/**
 * Compute the control action, stores in the json file
 * provided to it and returns a status code (1 = success).
 * @param   controlJson   file pointer to the output json file
 * @return  status        code
 */
uint_t SmpcController::controlAction(fstream& controlOutputJson){
	if( controlOutputJson.is_open()){
		uint_t status;
		uint_t nu = ptrMySmpcConfig->getNU();
		real_t *currentControl = new real_t[nu];
		size_t initialFreeBytes;
		size_t totalBytes;
		size_t finalFreeBytes;
		_CUDA( hipMemGetInfo(&initialFreeBytes, &totalBytes) );
		ptrMyEngine->updateStateControl(ptrMySmpcConfig->getCurrentX(), ptrMySmpcConfig->getPrevU(),
				ptrMySmpcConfig->getPrevDemand());
		ptrMyEngine->eliminateInputDistubanceCoupling(ptrMyForecaster->getNominalDemand(),
				ptrMyForecaster->getNominalPrices());
		status = algorithmApg();
		_CUDA( hipMemcpy(devControlAction, devVecU, nu*sizeof(real_t),
				hipMemcpyDeviceToDevice) );
		projectionBox<<<1, nu>>>(devControlAction, ptrMyEngine->getSysUmin(), ptrMyEngine->getSysUmax(), nu, 0, nu);
		_CUDA( hipMemcpy(currentControl, devControlAction, nu*sizeof(real_t), hipMemcpyDeviceToHost));
		controlOutputJson << "\"control\" : [" ;
		for(uint_t iControl = 0; iControl < nu; iControl++ ){
			//cout << currentControl[iControl] << " " ;
			controlOutputJson << currentControl[iControl] << ", ";
		}
		//cout << endl;
		_CUDA( hipMemGetInfo(&finalFreeBytes, &totalBytes) );
		controlOutputJson << "]" << endl;
		delete [] currentControl;
		if( abs(finalFreeBytes - initialFreeBytes) > 0 ){
			cout << "RUNTIME ERROR: MEMORY LEAKS" << endl;
			return 0;
		}else
			return status;
	}else
		return 0;
}

/*
 * During the closed-loop of the controller,
 * the controller moves to the next time instance. It checks
 * for the flag SIMULATOR_FLAG, 1 corresponds to an in-build
 * simulator call given by `updateSmpcConfiguration()` and
 * 0 corresponds to external simulator.
 *
 * Reads the smpcControlConfiguration file for currentState,
 * previousDemand and previousControl action.
 */
void SmpcController::moveForewardInTime(){
	if(simulatorFlag){
		//compute get the control from the devControl, apply the projection,
		// compute x+Bu+e to get updated state
		uint_t nx = this->ptrMyEngine->getDwnNetwork()->getNumTanks();
		uint_t nu = this->ptrMyEngine->getDwnNetwork()->getNumControls();
		uint_t nd = this->ptrMyEngine->getDwnNetwork()->getNumDemands();
		real_t *previousControl = new real_t[nu];
		real_t *stateUpdate = new real_t[nx];
		real_t *previousDemand;
		real_t alpha = 1;

		//x = p
		_CUDA( hipMemcpy( devStateUpdate, ptrMyEngine->getVecCurrentState(), nx*sizeof(real_t),
				hipMemcpyDeviceToDevice) );
		// x = x+w
		_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nx, &alpha, ptrMyEngine->getVecE(), 1, devVecX, 1));
		// x = x+Bu
		_CUBLAS(hipblasSgemv(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, nx, nu, &alpha,
				ptrMyEngine->getSysMatB(), nx, devControlAction, 1, &alpha, devStateUpdate, 1) );
		_CUDA( hipMemcpy(stateUpdate, devStateUpdate, nx*sizeof(real_t), hipMemcpyDeviceToHost) );
		_CUDA( hipMemcpy(previousControl, devControlAction, nu*sizeof(real_t), hipMemcpyDeviceToHost) );
		previousDemand = this->ptrMyForecaster->getNominalDemand();
		//updateSmpcConfiguration(stateUpdate, previousControl, previousDemand);
		updateKpi( stateUpdate, previousControl );
		this->ptrMySmpcConfig->setCurrentState( stateUpdate );
		this->ptrMySmpcConfig->setPreviousControl( previousControl );
		this->ptrMySmpcConfig->setpreviousdemand( previousDemand );
		delete [] previousControl;
		delete [] stateUpdate;
		previousControl = NULL;
		stateUpdate = NULL;
		//previousDemand = NULL;
	}else{
		this->ptrMySmpcConfig->setCurrentState();
		this->ptrMySmpcConfig->setPreviousControl();
		this->ptrMySmpcConfig->setPreviousDemand();
	}
}

/**
 * Update tje json file using the commands from rapidJson functions
 * When the SIMULATOR FLAG is set to 1, the previousControl,
 * currentState and previousDemand vectors in the smpc controller
 * configuration file are set.
 */
void SmpcController::updateSmpcConfiguration(real_t* updateState,
		real_t* control,
		real_t* demand){
	//const char* fileName = ptrMySmpcConfig->getPathToControllerConfig().c_str();
	string pathToFileString = "../systemData/testControl.json";
	const char* fileName = pathToFileString.c_str();
	rapidjson::Document jsonDocument;
	//rapidjson::Value valueJson;
	uint_t nx = ptrMySmpcConfig->getNX();
	uint_t nu = ptrMySmpcConfig->getNU();
	uint_t nd = ptrMySmpcConfig->getND();
	FILE* infile = fopen(fileName, "r");
	char* readBuffer = new char[65536];
	rapidjson::FileReadStream configurationJsonStream(infile, readBuffer, sizeof(readBuffer));
	jsonDocument.ParseStream(configurationJsonStream);
	//jsonDocument.RemoveMember(VARNAME_CURRENT_X);
	//jsonDocument.RemoveMember(VARNAME_PREV_U);
	//jsonDocument.RemoveMember(VARNAME_PREV_DEMAND);

	rapidjson::Value currentXjson(rapidjson::kArrayType);
	rapidjson::Value previousUjson(rapidjson::kArrayType);
	rapidjson::Value previousDemandJson(rapidjson::kArrayType);
	rapidjson::Document::AllocatorType& allocator = jsonDocument.GetAllocator();

	for(uint_t iSize = 0; iSize < nx; iSize++){
		currentXjson.PushBack(rapidjson::Value().SetFloat( updateState[iSize] ), allocator);
		cout << updateState[iSize] << " ";
	}
	cout << endl;
	for(uint_t iSize = 0; iSize < nu; iSize++){
		previousUjson.PushBack(rapidjson::Value().SetFloat( control[iSize] ), allocator);
	}
	for(uint_t iSize = 0; iSize < nd; iSize++){
		previousDemandJson.PushBack(rapidjson::Value().SetFloat( demand[iSize] ), allocator);
	}
	jsonDocument.AddMember(VARNAME_CURRENT_X, currentXjson, jsonDocument.GetAllocator());
	jsonDocument.AddMember(VARNAME_PREV_U, previousUjson, jsonDocument.GetAllocator());
	jsonDocument.AddMember(VARNAME_PREV_DEMAND, previousDemandJson, jsonDocument.GetAllocator());

	FILE* outfile = fopen(fileName, "w");
	char* writeBuffer = new char[65536];
	rapidjson::FileWriteStream os(outfile, writeBuffer, sizeof(writeBuffer));

	rapidjson::Writer<rapidjson::FileWriteStream> writer(os);
	jsonDocument.Accept(writer);
	fclose(outfile);
	delete [] readBuffer;
	delete [] writeBuffer;
}

/**
 * update the KPI at the current time instance
 */
void SmpcController::updateKpi(real_t* state, real_t* control){
	uint_t nx = this->ptrMySmpcConfig->getNX();
	uint_t nu = this->ptrMySmpcConfig->getNU();

	real_t *safeX = this->ptrMyEngine->getDwnNetwork()->getXsafe();
	real_t *constantPrice = this->ptrMyEngine->getDwnNetwork()->getAlpha();
	real_t *variablePrice = this->ptrMyForecaster->getNominalPrices();
	real_t *previousControl = this->ptrMySmpcConfig->getPrevU();
	real_t weightEconomic = ptrMySmpcConfig->getWeightEconomical();
	real_t *deltaU = new real_t[nu];
	real_t *waterLevel = new real_t[nx];

	real_t ecoKpi = 0;
	real_t smKpi = 0;
	real_t saKpi = 0;
	real_t netKpi = 0;
	real_t safeValue = 0;

	for(uint_t iSize = 0; iSize < nu; iSize++){
		ecoKpi = ecoKpi + weightEconomic*(constantPrice[iSize] + variablePrice[iSize])*abs(control[iSize]);
		deltaU[iSize] = previousControl[iSize] - control[iSize];
		smKpi = smKpi + deltaU[iSize]*deltaU[iSize];
	}
	for(uint_t iSize = 0; iSize < nx; iSize++){
		waterLevel[iSize] = state[iSize] - safeX[iSize];
		if( waterLevel[iSize] > 0 ){
			waterLevel[iSize] = 0;
		}
		safeValue = safeValue + abs( safeX[iSize] );
		saKpi = saKpi + abs( waterLevel[iSize] );
		netKpi = netKpi + abs( state[iSize] );
	}

	economicKpi = economicKpi + ecoKpi;
	smoothKpi = smoothKpi + smKpi;
	safeKpi = safeKpi + saKpi;
	networkKpi = networkKpi + netKpi;
	//cout << saKpi << " "<< netKpi << " " << safeValue << endl;
	delete [] deltaU;
	delete [] waterLevel;
}

/*
 * Get the economical KPI upto the simulation horizon
 * @param    simualtionTime  simulation horizon
 */
real_t SmpcController::getEconomicKpi( uint_t simulationTime){
	real_t economicValue = economicKpi/(3600);
	return economicValue/simulationTime;
}

/*
 * Get the smooth KPI upto the simulation horizon
 * @param    simulationTime   simulation horizon
 */
real_t SmpcController::getSmoothKpi( uint_t simulationTime){
	real_t smoothValue = smoothKpi/(3600);
	return smoothValue/simulationTime;
}

/*
 * Get the  network KPI upto the simulation horizon
 * @param   simulationTime    simulation horizon
 */
real_t SmpcController::getNetworkKpi( uint_t simulationTime){
	real_t networkKpiTime = networkKpi;
	real_t safeLevelNorm = 0;
	uint_t nx = this->ptrMySmpcConfig->getNX();
	for(uint_t iSize = 0; iSize < nx; iSize++){
		safeLevelNorm = safeLevelNorm + this->getDwnNetwork()->getXsafe()[iSize];
	}
	networkKpiTime = 100*simulationTime*safeLevelNorm/networkKpiTime;
	return networkKpiTime;
}

/*
 * Get the safety KPI upto the simulation horizon
 * @param   simulationTime    simulation horizon
 */
real_t SmpcController::getSafetyKpi( uint_t simulationTime){
	return safeKpi;
}

/**
 * Get's the network object
 * @return  DwnNetwork
 */
DwnNetwork* SmpcController::getDwnNetwork(){
	return ptrMyEngine->getDwnNetwork();
}
/**
 * Get's the scenario tree object
 * @return scenarioTree
 */
ScenarioTree* SmpcController::getScenarioTree(){
	return ptrMyEngine->getScenarioTree();
}
/**
 * Get's the forecaster object
 * @return Forecaster
 */
Forecaster* SmpcController::getForecaster(){
	return ptrMyForecaster;
}
/**
 * Get's the Smpc controller configuration object
 * @return SmpcConfiguration
 */
SmpcConfiguration* SmpcController::getSmpcConfiguration(){
	return ptrMySmpcConfig;
}
/**
 * Get's the Engine object
 * @return Engine
 */
Engine* SmpcController::getEngine(){
	return ptrMyEngine;
}

SmpcController::~SmpcController(){
	_CUDA( hipFree(devVecX) );
	_CUDA( hipFree(devVecU) );
	_CUDA( hipFree(devVecV) );
	_CUDA( hipFree(devVecXi) );
	_CUDA( hipFree(devVecPsi) );
	_CUDA( hipFree(devVecAcceleratedXi) );
	_CUDA( hipFree(devVecAcceleratedPsi) );
	_CUDA( hipFree(devVecPrimalXi) );
	_CUDA( hipFree(devVecPrimalPsi) );
	_CUDA( hipFree(devVecDualXi) );
	_CUDA( hipFree(devVecDualPsi) );
	_CUDA( hipFree(devVecUpdateXi) );
	_CUDA( hipFree(devVecUpdatePsi) );
	_CUDA( hipFree(devVecPrimalInfeasibilty) );
	_CUDA( hipFree(devVecQ) );
	_CUDA( hipFree(devVecR) );
	_CUDA( hipFree(devControlAction) );
	_CUDA( hipFree(devStateUpdate) );

	_CUDA( hipFree(devPtrVecX) );
	_CUDA( hipFree(devPtrVecU) );
	_CUDA( hipFree(devPtrVecV) );
	_CUDA( hipFree(devPtrVecAcceleratedXi) );
	_CUDA( hipFree(devPtrVecAcceleratedPsi) );
	_CUDA( hipFree(devPtrVecPrimalXi) );
	_CUDA( hipFree(devPtrVecPrimalPsi) );
	_CUDA( hipFree(devPtrVecQ));
	_CUDA( hipFree(devPtrVecR));

	free(vecPrimalInfs);
	devVecX = NULL;
	devVecU = NULL;
	devVecV = NULL;
	devVecXi = NULL;
	devVecPsi = NULL;
	devVecAcceleratedXi = NULL;
	devVecAcceleratedPsi = NULL;
	devVecPrimalXi = NULL;
	devVecPrimalPsi = NULL;
	devVecDualXi = NULL;
	devVecDualPsi = NULL;
	devVecUpdateXi = NULL;
	devVecUpdatePsi = NULL;
	devVecPrimalInfeasibilty = NULL;
	devVecQ = NULL;
	devVecR = NULL;
	devControlAction = NULL;
	devStateUpdate = NULL;

	devPtrVecX = NULL;
	devPtrVecU = NULL;
	devPtrVecV = NULL;
	devPtrVecAcceleratedXi = NULL;
	devPtrVecAcceleratedPsi = NULL;
	devPtrVecPrimalXi = NULL;
	devPtrVecPrimalPsi = NULL;
	devPtrVecQ = NULL;
	devPtrVecR = NULL;

	vecPrimalInfs = NULL;
}
