#include "hip/hip_runtime.h"
#include <cuda_device_runtime_api.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "rapidjson/document.h"
#include "rapidjson/rapidjson.h"
#include "rapidjson/filereadstream.h"
#include "DefinitionHeader.h"
#include "Engine.cuh"
//#include "cudaKernelHeader.cuh"

/*TODO REMOVE these type definitions from here - they are already defined in
		   DefinitionHeader.cuh (don't forget to rename DefinitionHeader.cuh into
		   Configuration.cuh.) */



Engine::Engine(DWNnetwork *myNetwork, Forecaster *myForecaster, unitTest *myTestor){
	cout << "allocating memory for the engine \n";
	ptrMyNetwork = myNetwork;
	ptrMyForecaster = myForecaster;
	ptrMyTestor = myTestor;
	uint_t nx = ptrMyNetwork->NX;
	uint_t nu = ptrMyNetwork->NU;
	uint_t nv = ptrMyNetwork->NV;
	uint_t nodes = ptrMyForecaster->nNodes;
	allocateSystemDevice();
	allocateForecastDevice();
	hipblasCreate(&handle);
	_CUDA( hipMalloc((void**)&devMatPhi, 2*nodes*nv*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatPsi, nodes*nu*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatTheta, nodes*nx*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatOmega, nodes*nv*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatSigma, nodes*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatD, 2*nodes*nv*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatF, nodes*nv*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatG, nodes*nv*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecUhat, nodes*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecBeta, nodes*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecE, nodes*nx*sizeof(real_t)) );

	_CUDA( hipMalloc((void**)&devPtrMatPhi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatPsi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatTheta, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatOmega, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatSigma, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatD, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatF, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatG, nodes*sizeof(real_t*)) );

	real_t** ptrMatPhi = new real_t*[nodes];
	real_t** ptrMatPsi = new real_t*[nodes];
	real_t** ptrMatTheta = new real_t*[nodes];
	real_t** ptrMatSigma = new real_t*[nodes];
	real_t** ptrMatOmega = new real_t*[nodes];
	real_t** ptrMatD = new real_t*[nodes];
	real_t** ptrMatF = new real_t*[nodes];
	real_t** ptrMatG = new real_t*[nodes];

	for(int i = 0;i < nodes; i++){
		ptrMatPhi[i] = &devMatPhi[2*i*nv*nx];
		ptrMatPsi[i] = &devMatPsi[i*nv*nu];
		ptrMatTheta[i] = &devMatTheta[i*nx*nv];
		ptrMatOmega[i] = &devMatOmega[i*nv*nv];
		ptrMatSigma[i] = &devMatSigma[i*nv];
		ptrMatD[i] = &devMatD[2*i*nv*nx];
		ptrMatF[i] = &devMatF[i*nv*nu];
		ptrMatG[i] = &devMatG[i*nv*nx];
	}

	_CUDA( hipMemcpy(devPtrMatPhi, ptrMatPhi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatPsi, ptrMatPsi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatTheta, ptrMatTheta, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatSigma, ptrMatSigma, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatOmega, ptrMatOmega, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatD, ptrMatD, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatF, ptrMatF, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatG, ptrMatG, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );

	delete [] ptrMatPhi;
	delete [] ptrMatPsi;
	delete [] ptrMatTheta;
	delete [] ptrMatSigma;
	delete [] ptrMatOmega;
	delete [] ptrMatD;
	delete [] ptrMatF;
	delete [] ptrMatG;
}

void Engine::allocateForecastDevice(){
	uint_t nodes = ptrMyForecaster->nNodes;
	uint_t N = ptrMyForecaster->N;
	uint_t K = ptrMyForecaster->K;
	uint_t ND = ptrMyNetwork->ND;
	uint_t N_NONLEAF_NODES = ptrMyForecaster->nNonleafNodes;
	_CUDA( hipMalloc((void**)&devTreeStages, nodes*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeNodesPerStage, (N + 1)*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeNodesPerStageCumul, (N + 2)*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeLeaves, K*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeNumChildren, N_NONLEAF_NODES*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeAncestor, nodes*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeNumChildrenCumul, nodes*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeProb, nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devTreeValue, nodes*ND*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devForecastValue, N*ND*sizeof(real_t)) );
}

void Engine::initialiseForecastDevice(){
	uint_t nodes = ptrMyForecaster->nNodes;
	uint_t N = ptrMyForecaster->N;
	uint_t K = ptrMyForecaster->K;
	uint_t ND = ptrMyNetwork->ND;
	uint_t N_NONLEAF_NODES = ptrMyForecaster->nNonleafNodes;
	_CUDA( hipMemcpy(devTreeStages, ptrMyForecaster->stages, nodes*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeNodesPerStage, ptrMyForecaster->nodesPerStage, (N + 1)*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeNodesPerStageCumul, ptrMyForecaster->nodesPerStageCumul, (N + 2)*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeLeaves, ptrMyForecaster->leaves, K*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeNumChildren, ptrMyForecaster->nChildren, N_NONLEAF_NODES*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeAncestor, ptrMyForecaster->ancestor, nodes*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeNumChildrenCumul, ptrMyForecaster->nChildrenCumul, nodes*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeProb, ptrMyForecaster->probNode, nodes*sizeof(real_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeValue, ptrMyForecaster->valueNode, nodes*ND*sizeof(real_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devForecastValue, ptrMyForecaster->dHat, N*ND*sizeof(real_t), hipMemcpyHostToDevice) );
}

void Engine::allocateSystemDevice(){
	uint_t nx = ptrMyNetwork->NX;
	uint_t nu = ptrMyNetwork->NU;
	uint_t nv = ptrMyNetwork->NV;
	uint_t nd = ptrMyNetwork->ND;
	uint_t nodes = ptrMyForecaster->nNodes;
	uint_t ns = ptrMyForecaster->K;
	_CUDA( hipMalloc((void**)&devSysMatB, ns*nx*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysMatL, ns*nv*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysMatLhat, ns*nu*nd*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysMatF, 2*nodes*nx*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysMatG, nodes*nu*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysXmin, nodes*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysXmax, nodes*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysXs, nodes*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysXsUpper, nodes*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysUmin, nodes*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysUmax, nodes*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysCostW, nodes*nv*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecCurrentState, nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPreviousControl, nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPreviousUhat, nu*sizeof(real_t)) );

	_CUDA( hipMalloc((void**)&devPtrSysMatB, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrSysMatL, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrSysMatLhat, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrSysMatF, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrSysMatG, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrSysCostW, nodes*sizeof(real_t*)) );

	real_t **ptrSysMatB = new real_t*[nodes];
	real_t **ptrSysMatL = new real_t*[nodes];
	real_t **ptrSysMatLhat = new real_t*[nodes];
	real_t **ptrSysMatF = new real_t*[nodes];
	real_t **ptrSysMatG = new real_t*[nodes];
	real_t **ptrSysCostW = new real_t*[nodes];

	for(int i = 0; i < nodes; i++ ){
		ptrSysMatF[i] = &devSysMatF[i*2*nx*nx];
		ptrSysMatG[i] = &devSysMatG[i*nu*nu];
		ptrSysCostW[i] = &devSysCostW[i*nv*nv];
	}
	for(int k = 0; k < ptrMyForecaster->N; k++){
		for(int j = 0; j < ptrMyForecaster->nodesPerStage[k]; j++){
			ptrSysMatB[ptrMyForecaster->nodesPerStageCumul[k] + j] = &devSysMatB[j*nx*nu];
			ptrSysMatL[ptrMyForecaster->nodesPerStageCumul[k] + j] = &devSysMatL[j*nu*nv];
			ptrSysMatLhat[ptrMyForecaster->nodesPerStageCumul[k] + j] = &devSysMatLhat[j*nu*nd];
		}
	}

	_CUDA( hipMemcpy(devPtrSysMatB, ptrSysMatB, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrSysMatL, ptrSysMatL, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrSysMatLhat, ptrSysMatLhat, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrSysMatF, ptrSysMatF, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrSysMatG, ptrSysMatG, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrSysCostW, ptrSysCostW, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );

	delete [] ptrSysMatB;
	delete [] ptrSysMatL;
	delete [] ptrSysMatLhat;
	delete [] ptrSysMatF;
	delete [] ptrSysMatG;
	delete [] ptrSysCostW;
}

void Engine::initialiseSystemDevice(){
	uint_t nodes = ptrMyForecaster->nNodes;
	uint_t nx = ptrMyNetwork->NX;
	uint_t nu = ptrMyNetwork->NU;
	uint_t nv = ptrMyNetwork->NV;
	uint_t ns = ptrMyForecaster->K;
	uint_t nd = ptrMyForecaster->dimDemand;
	uint_t N = ptrMyForecaster->N;
	uint_t numBlock, prevNodes;
	uint_t matFIdx, matGIdx;
	real_t *devMatDiagPrcnd;
	real_t *devCostMatW, *devMatvariable;
	real_t alpha = 1, beta = 0;

	//devSysXsUpper
	_CUDA( hipMalloc((void**)&devMatDiagPrcnd, N*(2*nx + nu)*sizeof(real_t)) );
	_CUDA( hipMemcpy(devMatDiagPrcnd, ptrMyNetwork->matDiagPrecnd, N*(2*nx + nu)*sizeof(real_t), hipMemcpyHostToDevice) );
	for (int iScen = 0; iScen < ns; iScen++){
		_CUDA( hipMemcpy(&devSysMatB[iScen*nx*nu], ptrMyNetwork->matB, nx*nu*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysMatL[iScen*nu*nv], ptrMyNetwork->matL, nu*nv*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysMatLhat[iScen*nu*nd], ptrMyNetwork->matLhat, nu*nd*sizeof(real_t), hipMemcpyHostToDevice) );
	}
	_CUDA( hipMalloc((void**)&devCostMatW, nu*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatvariable, nu*nv*sizeof(real_t)) );
	_CUDA( hipMemcpy(devCostMatW, ptrMyNetwork->matCostW, nu*nu*sizeof(real_t), hipMemcpyHostToDevice) );
	_CUBLAS( hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nu, nv, nu, &alpha, (const float*) devCostMatW, nu,
			(const float*) devSysMatL, nu, &beta, devMatvariable, nu) );
	_CUBLAS( hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nv, nv, nu, &alpha, (const float*) devSysMatL, nu,
			(const float*) devMatvariable, nu, &beta, devCostMatW, nv) );
	ptrMyTestor->checkObjectiveMatR(devCostMatW);

	_CUDA( hipMemset(devSysMatF, 0, nodes*2*nx*nx*sizeof(real_t)) );
	_CUDA( hipMemset(devSysMatG, 0, nodes*nu*nu*sizeof(real_t)) );
	for (int iStage = 0; iStage < N; iStage++){
		numBlock = ptrMyForecaster->nodesPerStage[iStage];
		prevNodes = ptrMyForecaster->nodesPerStageCumul[iStage];
		matFIdx = prevNodes*(2*nx * nx);
		matGIdx = prevNodes*(nu * nu);
		preconditionSystem<<<numBlock, 2*nx+nu>>>(&devSysMatF[matFIdx], &devSysMatG[matGIdx],
				&devMatDiagPrcnd[iStage*(2*nx + nu)], &devTreeProb[prevNodes], nx, nu );
	}
	/*
	_CUDA( hipMemcpy(y, devPtrSysMatF, nodes*sizeof(real_t*), hipMemcpyDeviceToHost) );

	for( int iNodes = 0; iNodes < nodes; iNodes++){
		//cout<< y[iNodes] << " ";
		_CUDA( hipMemcpy(x, &devSysMatF[iNodes*2*nx*nx], 2*nx*nx*sizeof(real_t), hipMemcpyDeviceToHost));
		for (int iRow = 0; iRow < 2*nx; iRow++){
			for(int iCol = 0; iCol < nx; iCol++){
				cout<< x[2*iCol*nx + iRow] << " ";
				//cout<< 2*iCol*nx + iRow << " ";
			}
			cout << "\n";
		}
	}

	_CUDA( hipMemcpy(x, devMatDiagPrcnd, N*(2*nx + nu)*sizeof(real_t), hipMemcpyDeviceToHost));
	for (int i = 0; i < N*(2*nx + nu); i++)
		cout<< x[i] << " " << i << " ";
	cout<<"\n";
	_CUDA( hipMemcpy(x, devTreeProb, N*sizeof(real_t), hipMemcpyDeviceToHost) );
	for (int i = 0; i < N; i++ )
		cout<< x[i] << " ";
	cout<<"\n";

	for( int iNodes = 0; iNodes < nodes; iNodes++){
		//cout<< y[iNodes] << " ";
		_CUDA( hipMemcpy(x, &devSysMatG[iNodes*nu*nu], nu*nu*sizeof(real_t), hipMemcpyDeviceToHost));

	}
	 */
	for (int iNodes = 0; iNodes < nodes; iNodes++){
		_CUDA( hipMemcpy(&devSysXmin[iNodes*nx], ptrMyNetwork->vecXmin, nx*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysXmax[iNodes*nx], ptrMyNetwork->vecXmax, nx*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysXs[iNodes*nx], ptrMyNetwork->vecXsafe, nx*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysUmin[iNodes*nx], ptrMyNetwork->vecUmin, nu*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysUmax[iNodes*nx], ptrMyNetwork->vecUmax, nu*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysCostW[iNodes*nv*nv], devCostMatW, nv*nv*sizeof(real_t), hipMemcpyDeviceToDevice) );
		_CUBLAS( hipblasSscal(handle, nx,&ptrMyForecaster->probNode[iNodes], &devSysXmax[iNodes*nx], 1) );
		_CUBLAS( hipblasSscal(handle, nx,&ptrMyForecaster->probNode[iNodes], &devSysXmin[iNodes*nx], 1) );
		_CUBLAS( hipblasSscal(handle, nx,&ptrMyForecaster->probNode[iNodes], &devSysXs[iNodes*nx], 1) );
		_CUBLAS( hipblasSscal(handle, nu,&ptrMyForecaster->probNode[iNodes], &devSysUmax[iNodes*nu], 1) );
		_CUBLAS( hipblasSscal(handle, nu,&ptrMyForecaster->probNode[iNodes], &devSysUmin[iNodes*nu], 1) );
		_CUBLAS( hipblasSscal(handle, nv*nv, &ptrMyForecaster->probNode[iNodes], &devSysCostW[iNodes*nv*nv], 1) );
	}
	//_CUDA(hipMemcpy(devSysXsUpper, devSysXmax, nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	uint_t scaleMax = pow(2, 7) - 1;
	_CUDA( hipMemset(devSysXsUpper, scaleMax, nx*nodes*sizeof(real_t)) );
	//_CUBLAS(hipblasSscal(handle, nx*nodes, &scaleMax, devSysXsUpper, 1));
	_CUDA( hipFree(devMatDiagPrcnd) );
	_CUDA( hipFree(devMatvariable) );
	_CUDA( hipFree(devCostMatW) );
}

void  Engine::factorStep(){
	real_t scale[2] = {-0.5, 1};
	real_t alpha = 1.0;
	real_t beta = 0.0;
	uint_t iStageCumulNodes, iStageNodes;
	real_t *devMatBbar, *devMatGbar;
	real_t **devPtrMatBbar, **devPtrMatGbar, **ptrMatBbar, **ptrMatGbar;
	uint_t ns = ptrMyForecaster->K;
	uint_t nx = ptrMyNetwork->NX;
	uint_t nu = ptrMyNetwork->NU;
	uint_t nv = ptrMyNetwork->NV;
	uint_t N = ptrMyForecaster->N;
	//real_t *x = new real_t[2*nodes*nu*nu];
	//real_t **y = new real_t*[nodes];
	_CUDA( hipMalloc((void**)&devMatBbar, nv*nx*ns*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatGbar, nu*nv*ns*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devPtrMatBbar, ns*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatGbar, ns*sizeof(real_t*)) );
	ptrMatBbar = new real_t*[ns];
	ptrMatGbar = new real_t*[ns];
	for(int i = 0; i < ns; i++){
		ptrMatBbar[i] = &devMatBbar[i*nx*nv];
		ptrMatGbar[i] = &devMatGbar[i*nu*nv];
	}
	_CUDA( hipMemcpy(devPtrMatGbar, ptrMatGbar, ns*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatBbar, ptrMatBbar, ns*sizeof(real_t*), hipMemcpyHostToDevice) );
	// Bbar'
	_CUBLAS( hipblasSgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, nv, nx, nu, &alpha, (const float**)devPtrSysMatL, nu,
			(const float**)devPtrSysMatB, nx, &beta, devPtrMatBbar, nv, ns));

	for(int iStage = N-1; iStage > -1; iStage--){
		iStageCumulNodes = ptrMyForecaster->nodesPerStageCumul[iStage];
		iStageNodes = ptrMyForecaster->nodesPerStage[iStage];
		// omega=(p_k\bar{R})^{-1}
		inverseBatchMat( &devPtrSysCostW[iStageCumulNodes], &devPtrMatOmega[iStageCumulNodes], nv, iStageNodes );
		// effinet_f=GBar
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, nv, nu, nu, &alpha, (const float**)devPtrSysMatL, nu,
				(const float**)&devPtrMatG[iStageCumulNodes], nu, &beta, &devPtrMatF[iStageCumulNodes], nv, iStageNodes) );
		// effinet_g=\bar{B}'
		_CUDA( hipMemcpy(&devMatG[nx*nv*iStageCumulNodes], devMatBbar, nx*nv*iStageNodes*sizeof(real_t), hipMemcpyDeviceToDevice));
		// effinet_d=\bar{B}'F'
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nv, 2*nx, nx, &alpha, (const float**)devPtrMatBbar, nv,
				(const float**)&devPtrSysMatF[iStageCumulNodes], 2*nx, &beta, &devPtrMatD[iStageCumulNodes], nv, iStageNodes));
		// phi=\omega \bar{B}'F'
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 2*nx, nv, &scale[0], (const float**)
				&devPtrMatOmega[iStageCumulNodes], nv, (const float**)&devPtrMatD[iStageCumulNodes], nv, &beta,
				&devPtrMatPhi[iStageCumulNodes], nv, iStageNodes));
		// theta=\omega \bar{B}'
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nv, nx, nv, &scale[0], (const float**)
				&devPtrMatOmega[iStageCumulNodes], nv, (const float**)devPtrMatBbar, nv, &beta,
				&devPtrMatTheta[iStageCumulNodes], nv , iStageNodes));
		// psi=\omega \bar{G}'
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nv, nu, nv, &scale[0],(const float**)
				&devPtrMatOmega[iStageCumulNodes], nv, (const float**)&devPtrMatF[iStageCumulNodes], nv, &beta,
				&devPtrMatPsi[iStageCumulNodes], nv, iStageNodes));
	}
	printf("Factor step is completed\n");

	delete [] ptrMatBbar;
	delete [] ptrMatGbar;
	_CUDA(hipFree(devMatBbar));
	_CUDA(hipFree(devMatGbar));
	_CUDA(hipFree(devPtrMatBbar));
	_CUDA(hipFree(devPtrMatGbar));
}

void Engine::eliminateInputDistubanceCoupling(){
	uint_t ns = ptrMyForecaster->K;
	uint_t nx = ptrMyNetwork->NX;
	uint_t nu = ptrMyNetwork->NU;
	uint_t nv = ptrMyNetwork->NV;
	uint_t nd = ptrMyNetwork->ND;
	uint_t N =  ptrMyForecaster->N;
	uint_t nodes = ptrMyForecaster->nNodes;
	uint_t numNonleafNodes = ptrMyForecaster->nNonleafNodes;
	real_t alpha = 1, beta = 0;
	real_t *devMatGd;
	real_t **devPtrMatGd, **devPtrVecE, **devPtrVecDemand;
	real_t **ptrMatGd = new real_t*[ns];
	real_t **ptrVecE = new real_t*[nodes];
	real_t **ptrVecDemand = new real_t*[nodes];
	real_t **ptrVecUhat = new real_t*[nodes];
	real_t *devVecDemand, *devVecDemandHat;
	real_t **devPtrVecUhat, *devVecDeltaUhat, *devVecZeta;
	real_t *devCostVecAlpha, *devCostVecAlpha1, *devVecAlphaBar;
	real_t *devMatRhat;
	uint_t iStageNodes, iStageCumulNodes, jNodes;

	_CUDA( hipMalloc((void**)&devVecDemand, nodes*nd*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDemandHat, N*nd*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatGd, ns*nx*nd*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devPtrMatGd, ns*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecE, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecDemand, nodes*sizeof(real_t*)));
	_CUDA( hipMalloc((void**)&devPtrVecUhat, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devCostVecAlpha, N*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devCostVecAlpha1, nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecAlphaBar, N*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatRhat, nu*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDeltaUhat, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecZeta, nu*nodes*sizeof(real_t)) );

	for (int iScenario = 0; iScenario < ns; iScenario++){
		_CUDA( hipMemcpy(&devMatGd[iScenario*nx*nd], ptrMyNetwork->matGd, nx*nd*sizeof(real_t), hipMemcpyHostToDevice) );
		ptrMatGd[iScenario] = &devMatGd[iScenario*nx*nd];
	}
	for( int iNode = 0; iNode < nodes; iNode++){
		ptrVecE[iNode] = &devVecE[iNode*nx];
		ptrVecDemand[iNode] = &devVecDemand[iNode*nd];
		ptrVecUhat[iNode] = &devVecUhat[iNode*nu];
	}
	_CUDA( hipMemcpy(devPtrMatGd, ptrMatGd, ns*sizeof(real_t*), hipMemcpyHostToDevice));
	_CUDA( hipMemcpy(devPtrVecE, ptrVecE, nodes*sizeof(real_t*), hipMemcpyHostToDevice));
	_CUDA( hipMemcpy(devPtrVecUhat, ptrVecUhat, nu*sizeof(real_t*), hipMemcpyHostToDevice));
	_CUDA( hipMemcpy(devVecDemand, ptrMyForecaster->valueNode, nodes*nd*sizeof(real_t), hipMemcpyHostToDevice ));
	_CUDA( hipMemcpy(devVecDemandHat, ptrMyForecaster->dHat, N*nd*sizeof(real_t), hipMemcpyHostToDevice ));
	// d(node) = dhat(stage) + d(node)
	for (int iStage = 0 ; iStage < N; iStage++){
		iStageCumulNodes = ptrMyForecaster->nodesPerStageCumul[iStage];
		iStageNodes = ptrMyForecaster->nodesPerStage[iStage];
		for(int j = 0; j < iStageNodes; j++){
			jNodes = iStageCumulNodes + j;
			_CUBLAS( hipblasSaxpy(handle, nd, &alpha, &devVecDemandHat[iStage*nd], 1, &devVecDemand[jNodes*nd],1) );
		}
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nx, 1, nd, &alpha, (const float**)
				&devPtrMatGd, nx, (const float**)devPtrVecDemand[iStageCumulNodes], nd, &beta,
				&devPtrVecE[iStageCumulNodes], nx , iStageNodes));
	}
	// uhat = Lhat*d
	_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nu, 1, nd, &alpha, (const float**)
			devPtrSysMatLhat, nu, (const float**)devPtrVecDemand, nd, &beta, devPtrVecUhat, nu , nodes));
	//uhat
	//alphaBar = L* (alpha1 +alpah2)
	_CUDA( hipMemcpy(devCostVecAlpha, ptrMyNetwork->vecCostAlpha2, N*nu*sizeof(real_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devCostVecAlpha1, ptrMyNetwork->vecCostAlpha1, nu*sizeof(real_t), hipMemcpyHostToDevice));
	for(int iStage = 0; iStage < N; iStage++){
		_CUBLAS( hipblasSaxpy(handle, N*nu, &alpha, devCostVecAlpha1, 1, &devCostVecAlpha[iStage*nu], 1) );
	}
	_CUBLAS( hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nv, N, nu, &alpha, (const float*) devSysMatL, nu,
			(const float*)devCostVecAlpha, nu, &beta, devVecAlphaBar, nv));
	_CUBLAS( hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nu, nv, nu, &alpha, (const float*) devSysCostW, nu,
			(const float*)devSysMatL, nu, &beta, devMatRhat, nu));
	// Beta
	calculateDiffUhat<<<nodes, nu>>>(devVecDeltaUhat, devVecUhat, devVecPreviousUhat, devTreeAncestor, nu, nodes);
	calculateZeta<<<nodes, nu>>>(devVecZeta, devVecDeltaUhat, devTreeProb, devTreeNumChildrenCumul, nu, numNonleafNodes, nodes);
	alpha = 2;
	_CUBLAS( hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nv, nodes, nu, &alpha, (const float *) devMatRhat, nu,
			(const float *) devVecZeta, nu, &beta, devVecBeta, nv) );
	alpha = 1;
	for(int iNode = 0; iNode < nodes; iNode++){
		real_t scale = ptrMyForecaster->probNode[iNode];
		_CUBLAS( hipblasSaxpy(handle, nv, &scale, &devVecAlphaBar[nv*iNode],1, &devVecBeta[nv*iNode],1) );
	}

	delete [] ptrMatGd;
	delete [] ptrVecE;
	delete [] ptrVecDemand;
	delete [] ptrVecUhat;
	_CUDA(hipFree(devVecDemand));
	_CUDA(hipFree(devVecDemandHat));
	_CUDA(hipFree(devMatGd));
	_CUDA(hipFree(devPtrMatGd));
	_CUDA(hipFree(devPtrVecE));
	_CUDA(hipFree(devPtrVecDemand));
	_CUDA(hipFree(devPtrVecUhat));
	_CUDA(hipFree(devCostVecAlpha));
	_CUDA(hipFree(devCostVecAlpha1));
	_CUDA(hipFree(devVecAlphaBar));
	_CUDA(hipFree(devMatRhat));
	_CUDA(hipFree(devVecDeltaUhat));
	_CUDA(hipFree(devVecZeta));
}

void Engine::updateStateControl(){
	_CUDA( hipMemcpy(devVecPreviousControl, ptrMyNetwork->prevU, ptrMyNetwork->NU*sizeof(real_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devVecPreviousUhat, ptrMyNetwork->prevUhat, ptrMyNetwork->NU*sizeof(real_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devVecCurrentState, ptrMyNetwork->currentX, ptrMyNetwork->NX*sizeof(real_t), hipMemcpyHostToDevice) );
}

void Engine::testStupidFunction(){
	real_t maxUpperbound = 0;
	uint_t nx = ptrMyNetwork->NX;
	_CUBLAS( hipblasSnrm2(handle, 1, devSysXsUpper, 1, &maxUpperbound) );
	cout<< maxUpperbound << endl;
	_CUBLAS( hipblasSnrm2(handle, nx, devSysXmax, 1, &maxUpperbound) );
	cout<< maxUpperbound << endl;
}

void Engine::inverseBatchMat(float** src, float** dst, int n, int batchSize){
	uint_t *P, *INFO;

	_CUDA(hipMalloc((void**)&P, n * batchSize * sizeof(uint_t)));
	_CUDA(hipMalloc((void**)&INFO, batchSize * sizeof(uint_t)));

	uint_t lda = n;

	real_t** x=(real_t**)malloc(batchSize*sizeof(real_t*));
	real_t* y=(real_t*)malloc(n*n*sizeof(real_t));


	_CUBLAS(hipblasSgetrfBatched(handle,n,src,lda,P,INFO,batchSize));

	uint_t INFOh[batchSize];

	_CUDA(hipMemcpy(INFOh,INFO,batchSize*sizeof(uint_t),hipMemcpyDeviceToHost));
	for (uint_t i = 0; i < batchSize; i++){
		if(INFOh[i] != 0)
		{
			fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
			hipDeviceReset();
			exit(EXIT_FAILURE);
		}
	}

	_CUBLAS(hipblasSgetriBatched(handle,n,(const float **)src,lda,P,dst,lda,INFO,batchSize));
	_CUDA(hipMemcpy(INFOh,INFO,batchSize*sizeof(uint_t),hipMemcpyDeviceToHost));

	for (uint_t i = 0; i < batchSize; i++)
		if(INFOh[i] != 0)
		{
			fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
			hipDeviceReset();
			exit(EXIT_FAILURE);
		}

	_CUDA(hipFree(P));
	_CUDA(hipFree(INFO));
}

void Engine::testInverse(){
	uint_t size_n= 3;
	uint_t batch_size=2;

	real_t* matA=(real_t*)malloc(size_n*size_n*batch_size*sizeof(real_t));
	real_t* inv_matA=(real_t*)malloc(size_n*size_n*batch_size*sizeof(real_t));
	real_t** ptr_matA=(real_t**)malloc(batch_size*sizeof(real_t*));
	real_t** ptr_inv_matA=(real_t**)malloc(batch_size*sizeof(real_t*));

	real_t *dev_matA,*dev_inv_matA,**dev_ptr_matA,**dev_ptr_inv_matA;

	_CUDA(hipMalloc((void**)&dev_matA,batch_size*size_n*size_n*sizeof(real_t)));
	_CUDA(hipMalloc((void**)&dev_inv_matA,batch_size*size_n*size_n*sizeof(real_t)));

	_CUDA(hipMalloc((void**)&dev_ptr_matA,batch_size*sizeof(real_t*)));
	_CUDA(hipMalloc((void**)&dev_ptr_inv_matA,batch_size*sizeof(real_t*)));

	real_t temp;
	for(int k=0;k<batch_size;k++){
		for(int i=0;i<size_n;i++)
			for(int j=0;j<size_n;j++){
				temp=(real_t)(rand() % 29)/32;
				matA[k*size_n*size_n+i*size_n+j]=temp;
				if(i==j){
					matA[k*size_n*size_n+i*size_n+j]=0.5;
				}
			}
		ptr_matA[k]=&dev_matA[k*size_n*size_n];
		ptr_inv_matA[k]=&dev_inv_matA[k*size_n*size_n];
	}
	for(int k=0;k<batch_size;k++){
		printf("matrix :%d \n",k);
		for(int i=0;i<size_n;i++){
			for(int j=0;j<size_n;j++){
				printf("%f ",matA[k*size_n*size_n+i*size_n+j]);
			}
			printf("\n");
		}
	}

	_CUDA(hipMemcpy(dev_matA,matA,batch_size*size_n*size_n*sizeof(real_t),hipMemcpyHostToDevice));
	_CUDA(hipMemcpy(dev_ptr_matA,ptr_matA,batch_size*sizeof(real_t*),hipMemcpyHostToDevice));

	_CUDA(hipMemcpy(dev_ptr_inv_matA,ptr_inv_matA,batch_size*sizeof(real_t*),hipMemcpyHostToDevice));


	this->inverseBatchMat(dev_ptr_matA,dev_ptr_inv_matA,size_n,batch_size);
	_CUDA(hipMemcpy(inv_matA,dev_inv_matA,batch_size*size_n*size_n*sizeof(real_t),hipMemcpyDeviceToHost));
	for(int k=0;k<batch_size;k++){
		printf("inverse of matrix :%d \n",k);
		for(int i=0;i<size_n;i++){
			for(int j=0;j<size_n;j++){
				printf("%f ",inv_matA[k*size_n*size_n+i*size_n+j]);
			}
			printf("\n");
		}
	}
	printf("Test successful\n");

	free(matA);
	free(ptr_matA);
	free(inv_matA);
	free(ptr_inv_matA);

	_CUDA(hipFree(dev_matA));
	_CUDA(hipFree(dev_inv_matA));
	_CUDA(hipFree(dev_ptr_matA));
	_CUDA(hipFree(dev_ptr_inv_matA));
}

void Engine::testPrecondtioningFunciton(){
	uint_t nx = ptrMyNetwork->NX;
	uint_t nu = ptrMyNetwork->NU;
	real_t *x = new real_t[2*nu*nu];
	real_t *devMatDiagPrcnd;
	_CUDA( hipMalloc((void**)&devMatDiagPrcnd, ptrMyForecaster->N*(2*nx + nu)*sizeof(real_t)) );
	_CUDA( hipMemcpy(devMatDiagPrcnd, ptrMyNetwork->matDiagPrecnd, ptrMyForecaster->N*(2*nx + nu)*sizeof(real_t), hipMemcpyHostToDevice) );

	preconditionSystem<<<2, 2*nx+nu>>>(&devSysMatF[0], &devSysMatG[0], &devMatDiagPrcnd[0], &devTreeProb[0], nx, nu );
	for (int i = 0; i < 2 ; i++ ){
		_CUDA( hipMemcpy(x, &devSysMatG[i*nu*nu], nu*nu*sizeof(real_t), hipMemcpyDeviceToHost));
		for (int iRow = 0; iRow < nu; iRow++){
			for(int iCol = 0; iCol < nu; iCol++){
				cout<< x[iCol*nu + iRow] << " ";
				//cout<< iCol*nu + iRow << " ";
			}
			cout << "\n";
		}
		_CUDA( hipMemcpy(x, &devSysMatF[i*2*nx*nx], 2*nx*nx*sizeof(real_t), hipMemcpyDeviceToHost));
		for (int iRow = 0; iRow < 2*nx; iRow++){
			for(int iCol = 0; iCol < nx; iCol++){
				cout<< x[2*iCol*nx + iRow] << " ";
				//cout<< 2*iCol*nx + iRow << " ";
			}
			cout << "\n";
		}
	}
	_CUDA( hipFree(devMatDiagPrcnd) );
	delete [] x;
}

void Engine::deallocateSystemDevice(){
	_CUDA( hipFree(devSysMatB) );
	_CUDA( hipFree(devSysMatL) );
	_CUDA( hipFree(devSysMatLhat) );
	_CUDA( hipFree(devSysMatF) );
	_CUDA( hipFree(devSysMatG) );
	_CUDA( hipFree(devSysXmin) );
	_CUDA( hipFree(devSysXmax) );
	_CUDA( hipFree(devSysXs) );
	_CUDA( hipFree(devSysXsUpper) );
	_CUDA( hipFree(devSysUmin) );
	_CUDA( hipFree(devSysUmax) );
	_CUDA( hipFree(devVecCurrentState));
	_CUDA( hipFree(devVecPreviousControl));
	_CUDA( hipFree(devVecPreviousUhat));

	_CUDA( hipFree(devPtrSysMatB) );
	_CUDA( hipFree(devPtrSysMatL) );
	_CUDA( hipFree(devPtrSysMatLhat) );
	_CUDA( hipFree(devPtrSysMatF) );
	_CUDA( hipFree(devPtrSysMatG) );
}

void Engine::deallocateForecastDevice(){
	_CUDA( hipFree(devTreeStages) );
	_CUDA( hipFree(devTreeNodesPerStage));
	_CUDA( hipFree(devTreeLeaves) );
	_CUDA( hipFree(devTreeNodesPerStageCumul) );
	_CUDA( hipFree(devTreeNumChildren) );
	_CUDA( hipFree(devTreeNumChildrenCumul) );
	_CUDA( hipFree(devTreeValue) );
	_CUDA( hipFree(devForecastValue) );
}
Engine::~Engine(){
	cout << "removing the memory of the engine \n";
	deallocateSystemDevice();
	deallocateForecastDevice();
	//delete ptrmyForecaster;
	//delete ptrMyNetwork;
	_CUDA(hipFree(devMatPhi));
	_CUDA(hipFree(devMatPsi));
	_CUDA(hipFree(devMatTheta));
	_CUDA(hipFree(devMatOmega));
	_CUDA(hipFree(devMatSigma));
	_CUDA(hipFree(devMatD));
	_CUDA(hipFree(devMatF));
	_CUDA(hipFree(devMatG));
	_CUDA(hipFree(devVecUhat));
	_CUDA(hipFree(devVecBeta));
	_CUDA(hipFree(devVecE));

	_CUDA(hipFree(devPtrMatPhi));
	_CUDA(hipFree(devPtrMatPsi));
	_CUDA(hipFree(devPtrMatTheta));
	_CUDA(hipFree(devPtrMatOmega));
	_CUDA(hipFree(devPtrMatSigma));
	_CUDA(hipFree(devPtrMatD));
	_CUDA(hipFree(devPtrMatF));
	_CUDA(hipFree(devPtrMatG));
	//_CUBLAS(hipblasDestroy(handle));
	hipblasDestroy(handle);
}
