#include "hip/hip_runtime.h"
/*
 *    GPU-accelerated scenario-based stochastic MPC for the operational
 *    management of drinking water networks.
 *    Copyright (C) 2017 Ajay. K. Sampathirao and P. Sopasakis
 *
 *    This library is free software; you can redistribute it and/or
 *    modify it under the terms of the GNU Lesser General Public
 *    License as published by the Free Software Foundation; either
 *    version 2.1 of the License, or (at your option) any later version.
 *
 *    This library is distributed in the hope that it will be useful,
 *    but WITHOUT ANY WARRANTY; without even the implied warranty of
 *    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 *    Lesser General Public License for more details.
 *
 *    You should have received a copy of the GNU Lesser General Public
 *    License along with this library; if not, write to the Free Software
 *    Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA 02110-1301 USA
 */

#include <cuda_device_runtime_api.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "rapidjson/document.h"
#include "rapidjson/rapidjson.h"
#include "rapidjson/filereadstream.h"
#include "Configuration.h"
#include "Engine.cuh"

Engine::Engine(DwnNetwork *myNetwork, ScenarioTree *myScenarioTree, SmpcConfiguration *mySmpcConfig){
	ptrMyNetwork = myNetwork;
	ptrMyScenarioTree = myScenarioTree;
	ptrMySmpcConfig = mySmpcConfig;
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	allocateSystemDevice();
	allocateScenarioTreeDevice();
	hipblasCreate(&handle);
	priceUncertaintyFlag = true;
	demandUncertaintyFlag = true;

	_CUDA( hipMalloc((void**)&devMatPhi, 2*nodes*nv*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatPsi, nodes*nu*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatTheta, nodes*nx*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatOmega, nodes*nv*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatSigma, nodes*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatD, 2*nodes*nv*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatF, nodes*nv*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatG, nodes*nv*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecUhat, nodes*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecBeta, nodes*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecE, nodes*nx*sizeof(real_t)) );

	_CUDA( hipMalloc((void**)&devPtrMatPhi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatPsi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatTheta, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatOmega, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatSigma, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatD, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatF, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatG, nodes*sizeof(real_t*)) );

	real_t** ptrMatPhi = new real_t*[nodes];
	real_t** ptrMatPsi = new real_t*[nodes];
	real_t** ptrMatTheta = new real_t*[nodes];
	real_t** ptrMatSigma = new real_t*[nodes];
	real_t** ptrMatOmega = new real_t*[nodes];
	real_t** ptrMatD = new real_t*[nodes];
	real_t** ptrMatF = new real_t*[nodes];
	real_t** ptrMatG = new real_t*[nodes];

	for(int i = 0;i < nodes; i++){
		ptrMatPhi[i] = &devMatPhi[2*i*nv*nx];
		ptrMatPsi[i] = &devMatPsi[i*nv*nu];
		ptrMatTheta[i] = &devMatTheta[i*nx*nv];
		ptrMatOmega[i] = &devMatOmega[i*nv*nv];
		ptrMatSigma[i] = &devMatSigma[i*nv];
		ptrMatD[i] = &devMatD[2*i*nv*nx];
		ptrMatF[i] = &devMatF[i*nv*nu];
		ptrMatG[i] = &devMatG[i*nv*nx];
	}

	_CUDA( hipMemcpy(devPtrMatPhi, ptrMatPhi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatPsi, ptrMatPsi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatTheta, ptrMatTheta, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatSigma, ptrMatSigma, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatOmega, ptrMatOmega, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatD, ptrMatD, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatF, ptrMatF, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatG, ptrMatG, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );

	delete [] ptrMatPhi;
	delete [] ptrMatPsi;
	delete [] ptrMatTheta;
	delete [] ptrMatSigma;
	delete [] ptrMatOmega;
	delete [] ptrMatD;
	delete [] ptrMatF;
	delete [] ptrMatG;
	ptrMatPhi = NULL;
	ptrMatPsi = NULL;
	ptrMatTheta = NULL;
	ptrMatSigma = NULL;
	ptrMatD = NULL;
	ptrMatF = NULL;
	ptrMatG = NULL;
}

Engine::Engine(SmpcConfiguration *smpcConfig){
	ptrMySmpcConfig = smpcConfig;
	string pathToNetwork = ptrMySmpcConfig->getPathToNetwork();
	string pathToScenarioTree = ptrMySmpcConfig->getPathToScenarioTree();
	ptrMyNetwork = new DwnNetwork( pathToNetwork );
	ptrMyScenarioTree = new ScenarioTree( pathToScenarioTree );

	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	allocateSystemDevice();
	allocateScenarioTreeDevice();
	hipblasCreate(&handle);
	priceUncertaintyFlag = true;
	demandUncertaintyFlag = true;

	_CUDA( hipMalloc((void**)&devMatPhi, 2*nodes*nv*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatPsi, nodes*nu*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatTheta, nodes*nx*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatOmega, nodes*nv*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatSigma, nodes*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatD, 2*nodes*nv*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatF, nodes*nv*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatG, nodes*nv*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecUhat, nodes*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecBeta, nodes*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecE, nodes*nx*sizeof(real_t)) );

	_CUDA( hipMalloc((void**)&devPtrMatPhi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatPsi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatTheta, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatOmega, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatSigma, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatD, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatF, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatG, nodes*sizeof(real_t*)) );

	real_t** ptrMatPhi = new real_t*[nodes];
	real_t** ptrMatPsi = new real_t*[nodes];
	real_t** ptrMatTheta = new real_t*[nodes];
	real_t** ptrMatSigma = new real_t*[nodes];
	real_t** ptrMatOmega = new real_t*[nodes];
	real_t** ptrMatD = new real_t*[nodes];
	real_t** ptrMatF = new real_t*[nodes];
	real_t** ptrMatG = new real_t*[nodes];

	for(int i = 0;i < nodes; i++){
		ptrMatPhi[i] = &devMatPhi[2*i*nv*nx];
		ptrMatPsi[i] = &devMatPsi[i*nv*nu];
		ptrMatTheta[i] = &devMatTheta[i*nx*nv];
		ptrMatOmega[i] = &devMatOmega[i*nv*nv];
		ptrMatSigma[i] = &devMatSigma[i*nv];
		ptrMatD[i] = &devMatD[2*i*nv*nx];
		ptrMatF[i] = &devMatF[i*nv*nu];
		ptrMatG[i] = &devMatG[i*nv*nx];
	}

	_CUDA( hipMemcpy(devPtrMatPhi, ptrMatPhi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatPsi, ptrMatPsi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatTheta, ptrMatTheta, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatSigma, ptrMatSigma, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatOmega, ptrMatOmega, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatD, ptrMatD, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatF, ptrMatF, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatG, ptrMatG, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );


	delete [] ptrMatPhi;
	delete [] ptrMatPsi;
	delete [] ptrMatTheta;
	delete [] ptrMatSigma;
	delete [] ptrMatOmega;
	delete [] ptrMatD;
	delete [] ptrMatF;
	delete [] ptrMatG;
	ptrMatPhi = NULL;
	ptrMatPsi = NULL;
	ptrMatTheta = NULL;
	ptrMatSigma = NULL;
	ptrMatD = NULL;
	ptrMatF = NULL;
	ptrMatG = NULL;
}

void Engine::allocateScenarioTreeDevice(){
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t N = ptrMyScenarioTree->getPredHorizon();
	uint_t K = ptrMyScenarioTree->getNumScenarios();
	uint_t ND = ptrMyNetwork->getNumDemands();
	uint_t NU = ptrMyNetwork->getNumControls();
	uint_t nNumNonLeafNodes = ptrMyScenarioTree->getNumNonleafNodes();
	_CUDA( hipMalloc((void**)&devTreeStages, nodes*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeNodesPerStage, (N + 1)*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeNodesPerStageCumul, (N + 2)*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeLeaves, K*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeNumChildren, nNumNonLeafNodes*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeAncestor, nodes*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeNumChildrenCumul, nodes*sizeof(uint_t)) );
	_CUDA( hipMalloc((void**)&devTreeProb, nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devTreeErrorDemand, nodes*ND*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devTreeErrorPrices, nodes*NU*sizeof(real_t)) );
	//_CUDA( hipMalloc((void**)&devForecastValue, N*ND*sizeof(real_t)) );
}

void Engine::initialiseScenarioTreeDevice(){
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t N = ptrMyScenarioTree->getPredHorizon();
	uint_t K = ptrMyScenarioTree->getNumScenarios();
	uint_t nd = ptrMyNetwork->getNumDemands();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t numNonLeafNodes = ptrMyScenarioTree->getNumNonleafNodes();
	_CUDA( hipMemcpy(devTreeStages, ptrMyScenarioTree->getStageNodes(), nodes*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeNodesPerStage, ptrMyScenarioTree->getNodesPerStage(), (N + 1)*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeNodesPerStageCumul, ptrMyScenarioTree->getNodesPerStageCumul(), (N + 2)*sizeof(uint_t),
			hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeLeaves, ptrMyScenarioTree->getLeaveArray(), K*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeNumChildren, ptrMyScenarioTree->getNumChildren(), numNonLeafNodes*sizeof(uint_t),
			hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeAncestor, ptrMyScenarioTree->getAncestorArray(), nodes*sizeof(uint_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeNumChildrenCumul, ptrMyScenarioTree->getNumChildrenCumul(), nodes*sizeof(uint_t),
			hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeProb, ptrMyScenarioTree->getProbArray(), nodes*sizeof(real_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeErrorDemand, ptrMyScenarioTree->getErrorDemandArray(), nodes*nd*sizeof(real_t),
			hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devTreeErrorPrices, ptrMyScenarioTree->getErrorPriceArray(), nodes*nu*sizeof(real_t),
				hipMemcpyHostToDevice) );
	//_CUDA( hipMemcpy(devForecastValue, ptrMyForecaster->dHat, N*ND*sizeof(real_t), hipMemcpyHostToDevice) );
}

void Engine::allocateSystemDevice(){
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t nd = ptrMyNetwork->getNumDemands();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t ns = ptrMyScenarioTree->getNumScenarios();
	uint_t N = ptrMyScenarioTree->getPredHorizon();
	uint_t *nodesPerStage = ptrMyScenarioTree->getNodesPerStage();
	uint_t *nodesPerStageCumul = ptrMyScenarioTree->getNodesPerStageCumul();
	uint_t iStage, iNode;
	_CUDA( hipMalloc((void**)&devSysMatB, ns*nx*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysMatL, ns*nv*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysMatLhat, ns*nu*nd*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysMatF, 2*nodes*nx*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysMatG, nodes*nu*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysXmin, nodes*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysXmax, nodes*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysXs, nodes*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysXsUpper, nodes*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysUmin, nodes*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysUmax, nodes*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devSysCostW, nodes*nv*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecCurrentState, nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPreviousControl, nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPreviousUhat, nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPreviousDemand, nd*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatWv, nu*nv*sizeof(real_t)) );

	_CUDA( hipMalloc((void**)&devPtrSysMatB, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrSysMatL, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrSysMatLhat, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrSysMatF, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrSysMatG, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrSysCostW, nodes*sizeof(real_t*)) );

	real_t **ptrSysMatB = new real_t*[nodes];
	real_t **ptrSysMatL = new real_t*[nodes];
	real_t **ptrSysMatLhat = new real_t*[nodes];
	real_t **ptrSysMatF = new real_t*[nodes];
	real_t **ptrSysMatG = new real_t*[nodes];
	real_t **ptrSysCostW = new real_t*[nodes];

	for(uint_t iNode = 0; iNode < nodes; iNode++ ){
		ptrSysMatF[iNode] = &devSysMatF[iNode*2*nx*nx];
		ptrSysMatG[iNode] = &devSysMatG[iNode*nu*nu];
		ptrSysCostW[iNode] = &devSysCostW[iNode*nv*nv];
	}
	for(iStage = 0; iStage < N; iStage++){
		for(iNode = 0; iNode < nodesPerStage[iStage]; iNode++){
			ptrSysMatB[nodesPerStageCumul[iStage] + iNode] = &devSysMatB[iNode*nx*nu];
			ptrSysMatL[nodesPerStageCumul[iStage] + iNode] = &devSysMatL[iNode*nu*nv];
			ptrSysMatLhat[nodesPerStageCumul[iStage] + iNode] = &devSysMatLhat[iNode*nu*nd];
		}
	}

	_CUDA( hipMemcpy(devPtrSysMatB, ptrSysMatB, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrSysMatL, ptrSysMatL, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrSysMatLhat, ptrSysMatLhat, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrSysMatF, ptrSysMatF, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrSysMatG, ptrSysMatG, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrSysCostW, ptrSysCostW, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );

	delete [] ptrSysMatB;
	delete [] ptrSysMatL;
	delete [] ptrSysMatLhat;
	delete [] ptrSysMatF;
	delete [] ptrSysMatG;
	delete [] ptrSysCostW;
	ptrSysMatB = NULL;
	ptrSysMatL = NULL;
	ptrSysMatLhat = NULL;
	ptrSysMatF = NULL;
	ptrSysMatG = NULL;
	ptrSysCostW = NULL;
	nodesPerStage = NULL;
	nodesPerStageCumul = NULL;
}

void Engine::initialiseSystemDevice(){
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nd = ptrMyNetwork->getNumDemands();
	uint_t ns = ptrMyScenarioTree->getNumScenarios();
	uint_t N = ptrMyScenarioTree->getPredHorizon();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t *nodesPerStage = ptrMyScenarioTree->getNodesPerStage();
	uint_t *nodesPerStageCumul = ptrMyScenarioTree->getNodesPerStageCumul();
	uint_t numBlock, prevNodes;
	uint_t matFIdx, matGIdx;
	uint_t stateIdx, controlIdx;
	real_t *devMatDiagPrcnd;
	real_t *devCostMatW, *devMatvariable;
	real_t alpha = 1, beta = 0;

	_CUDA( hipMalloc((void**)&devMatDiagPrcnd, N*(2*nx + nu)*sizeof(real_t)) );
	_CUDA( hipMemcpy(devMatDiagPrcnd, ptrMySmpcConfig->getMatPrcndDiag(), N*(2*nx + nu)*sizeof(real_t),
			hipMemcpyHostToDevice) );
	for (uint_t iScen = 0; iScen < ns; iScen++){
		_CUDA( hipMemcpy(&devSysMatB[iScen*nx*nu], ptrMyNetwork->getMatB(), nx*nu*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysMatL[iScen*nu*nv], ptrMySmpcConfig->getMatL(), nu*nv*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysMatLhat[iScen*nu*nd], ptrMySmpcConfig->getMatLhat(), nu*nd*sizeof(real_t),
				hipMemcpyHostToDevice) );
	}
	_CUDA( hipMalloc((void**)&devCostMatW, nu*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatvariable, nu*nv*sizeof(real_t)) );
	_CUDA( hipMemcpy(devCostMatW, ptrMySmpcConfig->getCostW(), nu*nu*sizeof(real_t), hipMemcpyHostToDevice) );
	_CUBLAS( hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nu, nv, nu, &alpha, (const real_t*) devCostMatW, nu,
			(const real_t*) devSysMatL, nu, &beta, devMatvariable, nu) );
	_CUDA( hipMemcpy( devMatWv, devMatvariable, nu*nv*sizeof(real_t), hipMemcpyDeviceToDevice) );
	_CUBLAS( hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nv, nv, nu, &alpha, (const real_t*) devSysMatL, nu,
			(const real_t*) devMatvariable, nu, &beta, devCostMatW, nv) );

	_CUDA( hipMemset(devSysMatF, 0, nodes*2*nx*nx*sizeof(real_t)) );
	_CUDA( hipMemset(devSysMatG, 0, nodes*nu*nu*sizeof(real_t)) );

	for (uint_t iNodes = 0; iNodes < nodes; iNodes++){
		_CUDA( hipMemcpy(&devSysXmin[iNodes*nx], ptrMyNetwork->getXmin(), nx*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysXmax[iNodes*nx], ptrMyNetwork->getXmax(), nx*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysXs[iNodes*nx], ptrMyNetwork->getXsafe(), nx*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysUmin[iNodes*nu], ptrMyNetwork->getUmin(), nu*sizeof(real_t), hipMemcpyHostToDevice) );
		_CUDA( hipMemcpy(&devSysUmax[iNodes*nu], ptrMyNetwork->getUmax(), nu*sizeof(real_t), hipMemcpyHostToDevice) );
		/*_CUBLAS( hipblasSscal(handle, nx, &ptrMyScenarioTree->getProbArray()[iNodes], &devSysXmax[iNodes*nx], 1) );
			_CUBLAS( hipblasSscal(handle, nx, &ptrMyScenarioTree->getProbArray()[iNodes], &devSysXmin[iNodes*nx], 1) );
			_CUBLAS( hipblasSscal(handle, nx, &ptrMyScenarioTree->getProbArray()[iNodes], &devSysXs[iNodes*nx], 1) );
			_CUBLAS( hipblasSscal(handle, nu, &ptrMyScenarioTree->getProbArray()[iNodes], &devSysUmax[iNodes*nu], 1) );
			_CUBLAS( hipblasSscal(handle, nu, &ptrMyScenarioTree->getProbArray()[iNodes], &devSysUmin[iNodes*nu], 1) );*/
		_CUDA( hipMemcpy(&devSysCostW[iNodes*nv*nv], devCostMatW, nv*nv*sizeof(real_t), hipMemcpyDeviceToDevice) );
		_CUBLAS( hipblasSscal(handle, nv*nv, &ptrMyScenarioTree->getProbArray()[iNodes], &devSysCostW[iNodes*nv*nv], 1) );
	}

	for (uint_t iStage = 0; iStage < N; iStage++){
		numBlock = nodesPerStage[iStage];
		prevNodes = nodesPerStageCumul[iStage];
		matFIdx = prevNodes*(2*nx * nx);
		matGIdx = prevNodes*(nu * nu);
		stateIdx = prevNodes*nx;
		controlIdx = prevNodes*nu;
		preconditionSystem<<<numBlock, 2*nx+nu>>>(&devSysMatF[matFIdx], &devSysMatG[matGIdx],
				&devMatDiagPrcnd[iStage*(2*nx + nu)], &devTreeProb[prevNodes], nx, nu );
		preconditionConstraintU<<<numBlock, nu>>>(&devSysUmax[controlIdx], &devSysUmin[controlIdx],
				&devMatDiagPrcnd[iStage*(2*nx + nu)], &devTreeProb[prevNodes], nu, numBlock);
		preconditionConstraintX<<<numBlock, nx>>>(&devSysXmax[stateIdx], &devSysXmin[stateIdx], &devSysXs[stateIdx],
				&devMatDiagPrcnd[iStage*(2*nx + nu) + nu], &devTreeProb[prevNodes], nx, numBlock);
	}

	//_CUDA(hipMemcpy(devSysXsUpper, devSysXmax, nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	uint_t scaleMax = pow(2, 7) - 1;
	_CUDA( hipMemset(devSysXsUpper, scaleMax, nx*nodes*sizeof(real_t)) );
	//_CUBLAS(hipblasSscal(handle, nx*nodes, &scaleMax, devSysXsUpper, 1));
	_CUDA( hipFree(devMatDiagPrcnd) );
	_CUDA( hipFree(devMatvariable) );
	_CUDA( hipFree(devCostMatW) );
	devMatDiagPrcnd = NULL;
	devMatvariable = NULL;
	devCostMatW = NULL;
}

void  Engine::factorStep(){
	initialiseScenarioTreeDevice();
	initialiseSystemDevice();
	real_t scale[2] = {-0.5, 1};
	real_t alpha = 1.0;
	real_t beta = 0.0;
	uint_t iStageCumulNodes, iStageNodes;
	real_t *devMatBbar, *devMatGbar;
	real_t **devPtrMatBbar, **devPtrMatGbar, **ptrMatBbar, **ptrMatGbar;
	uint_t ns = ptrMyScenarioTree->getNumScenarios();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t N = ptrMyScenarioTree->getPredHorizon();
	uint_t *nodesPerStage = ptrMyScenarioTree->getNodesPerStage();
	uint_t *nodesPerStageCumul = ptrMyScenarioTree->getNodesPerStageCumul();

	_CUDA( hipMalloc((void**)&devMatBbar, nv*nx*ns*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatGbar, nu*nv*ns*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devPtrMatBbar, ns*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrMatGbar, ns*sizeof(real_t*)) );
	ptrMatBbar = new real_t*[ns];
	ptrMatGbar = new real_t*[ns];
	for(uint_t i = 0; i < ns; i++){
		ptrMatBbar[i] = &devMatBbar[i*nx*nv];
		ptrMatGbar[i] = &devMatGbar[i*nu*nv];
	}
	_CUDA( hipMemcpy(devPtrMatGbar, ptrMatGbar, ns*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrMatBbar, ptrMatBbar, ns*sizeof(real_t*), hipMemcpyHostToDevice) );
	// Bbar'
	_CUBLAS( hipblasSgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, nv, nx, nu, &alpha, (const real_t**)devPtrSysMatL, nu,
			(const real_t**)devPtrSysMatB, nx, &beta, devPtrMatBbar, nv, ns));

	for(uint_t iStage = N-1; iStage > -1; iStage--){
		iStageCumulNodes = nodesPerStageCumul[iStage];
		iStageNodes = nodesPerStage[iStage];
		// omega=(p_k\bar{R})^{-1}
		inverseBatchMat( &devPtrSysCostW[iStageCumulNodes], &devPtrMatOmega[iStageCumulNodes], nv, iStageNodes );
		// effinet_f=GBar
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, nv, nu, nu, &alpha, (const real_t**)devPtrSysMatL, nu,
				(const real_t**)&devPtrSysMatG[iStageCumulNodes], nu, &beta, &devPtrMatF[iStageCumulNodes], nv, iStageNodes) );
		// effinet_g=\bar{B}'
		_CUDA( hipMemcpy(&devMatG[nx*nv*iStageCumulNodes], devMatBbar, nx*nv*iStageNodes*sizeof(real_t), hipMemcpyDeviceToDevice));
		// effinet_d=\bar{B}'F'
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nv, 2*nx, nx, &alpha, (const real_t**)devPtrMatBbar, nv,
				(const real_t**)&devPtrSysMatF[iStageCumulNodes], 2*nx, &beta, &devPtrMatD[iStageCumulNodes], nv, iStageNodes));
		// phi=\omega \bar{B}'F'
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 2*nx, nv, &scale[0], (const real_t**)
				&devPtrMatOmega[iStageCumulNodes], nv, (const real_t**)&devPtrMatD[iStageCumulNodes], nv, &beta,
				&devPtrMatPhi[iStageCumulNodes], nv, iStageNodes));
		// theta=\omega \bar{B}'
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nv, nx, nv, &scale[0], (const real_t**)
				&devPtrMatOmega[iStageCumulNodes], nv, (const real_t**)devPtrMatBbar, nv, &beta,
				&devPtrMatTheta[iStageCumulNodes], nv , iStageNodes));
		// psi=\omega \bar{G}'
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nv, nu, nv, &scale[0],(const real_t**)
				&devPtrMatOmega[iStageCumulNodes], nv, (const real_t**)&devPtrMatF[iStageCumulNodes], nv, &beta,
				&devPtrMatPsi[iStageCumulNodes], nv, iStageNodes));
	}
	//cout << "Factor step is completed\n";

	delete [] ptrMatBbar;
	delete [] ptrMatGbar;
	_CUDA(hipFree(devMatBbar));
	_CUDA(hipFree(devMatGbar));
	_CUDA(hipFree(devPtrMatBbar));
	_CUDA(hipFree(devPtrMatGbar));
	ptrMatBbar = NULL;
	ptrMatGbar = NULL;
	devMatBbar = NULL;
	devMatGbar = NULL;
	devPtrMatBbar = NULL;
	devPtrMatGbar = NULL;
}

/**
 *  pointer to the scenario tree
 */
ScenarioTree* Engine::getScenarioTree(){
	return ptrMyScenarioTree;
}
/**
 *  pointer to the DWN network
 */
DwnNetwork* Engine::getDwnNetwork(){
	return ptrMyNetwork;
}
/** ----GETTER'S FOR THE FACTOR STEP---*/
/**
 *  matrix Phi
 */
real_t* Engine::getMatPhi(){
	return devMatPhi;
}
/**
 * matrix Psi
 */
real_t* Engine::getMatPsi(){
	return devMatPsi;
}
/**
 * matrix Theta
 */
real_t* Engine::getMatTheta(){
	return devMatTheta;
}
/**
 * matrix Theta
 */
real_t* Engine::getMatOmega(){
	return devMatOmega;
}
/**
 * matrix Sigma
 */
real_t* Engine::getMatSigma(){
	return devMatSigma;
}
/**
 * matrix D
 */
real_t* Engine::getMatD(){
	return devMatD;
}
/**
 * matrix F (Factor step)
 */
real_t* Engine::getMatF(){
	return devMatF;
}
/**
 * matrix G (Factor step)
 */
real_t* Engine::getMatG(){
	return devMatG;
}
/**
 * pointer matrix Phi
 */
real_t** Engine::getPtrMatPhi(){
	return devPtrMatPhi;
}
/**
 * pointer matrix Psi
 */
real_t** Engine::getPtrMatPsi(){
	return devPtrMatPsi;
}
/**
 * pointer matrix Theta
 */
real_t** Engine::getPtrMatTheta(){
	return devPtrMatTheta;
}
/**
 * pointer matrix Omega
 */
real_t** Engine::getPtrMatOmega(){
	return devPtrMatOmega;
}
/**
 * pointer matrix Sigma
 */
real_t** Engine::getPtrMatSigma(){
	return devPtrMatSigma;
}
/**
 * pointer matrix D
 */
real_t** Engine::getPtrMatD(){
	return devPtrMatD;
}
/**
 * pointer matrix F (Factor step)
 */
real_t** Engine::getPtrMatF(){
	return devPtrMatF;
}
/**
 * pointer matrix G (Factor step)
 */
real_t** Engine::getPtrMatG(){
	return devPtrMatG;
}
/**
 * uhat
 */
real_t* Engine::getVecUhat(){
	return devVecUhat;
}
/**
 * beta control-distribution elimination
 */
real_t* Engine::getVecBeta(){
	return devVecBeta;
}
/**
 * e control-disturbance elimination
 */
real_t* Engine::getVecE(){
	return devVecE;
}
/** ---GETTER'S FOR THE SYSTEM MATRICES */
real_t* Engine::getSysMatB(){
	return devSysMatB;
}
/**
 * constraints matrix F
 */
real_t* Engine::getSysMatF(){
	return devSysMatF;
}
/**
 * constraints matrix G
 */
real_t* Engine::getSysMatG(){
	return devSysMatG;
}
/**
 * matrix L
 */
real_t* Engine::getSysMatL(){
	return devSysMatL;
}
/**
 * matrix Lhat
 */
real_t* Engine::getSysMatLhat(){
	return devSysMatLhat;
}
/**
 * pointer to Matrix B
 */
real_t** Engine::getPtrSysMatB(){
	return devPtrSysMatB;
}
/**
 * pointer to matrix F
 */
real_t** Engine::getPtrSysMatF(){
	return devPtrSysMatF;
}
/**
 * pointer to matrix G
 */
real_t** Engine::getPtrSysMatG(){
	return devPtrSysMatG;
}
/**
 * pointer to matrix L
 */
real_t** Engine::getPtrSysMatL(){
	return devPtrSysMatLhat;
}
/**
 * pointer to matrix Lhat
 */
real_t** Engine::getPtrSysMatLhat(){
	return devPtrSysMatLhat;
}
/**
 * previous control
 */
real_t* Engine::getVecPreviousControl(){
	return devVecPreviousControl;
}
/**
 * current state
 */
real_t* Engine::getVecCurrentState(){
	return devVecCurrentState;
}
/**
 * previous uhat
 */
real_t* Engine::getVecPreviousUhat(){
	return devVecPreviousUhat;
}
/**
 * previous demand
 */
real_t* Engine::getVecDemand(){
	return devVecPreviousDemand;
}
/** ----GETTER'S FOR THE SCENARIO TREE----*/
/**
 *  Array of the stage of the nodes at the tree
 */
uint_t* Engine::getTreeStages(){
	return devTreeStages;
}
/**
 * Array of nodes per stage
 */
uint_t* Engine::getTreeNodesPerStage(){
	return devTreeNodesPerStage;
}
/**
 * Array of past nodes
 */
uint_t* Engine::getTreeNodesPerStageCumul(){
	return devTreeNodesPerStageCumul;
}
/**
 * Array of the leaves
 */
uint_t* Engine::getTreeLeaves(){
	return devTreeLeaves;
}
/**
 * Array number of children
 */
uint_t* Engine::getTreeNumChildren(){
	return devTreeNumChildren;
}
/**
 * Array of ancestor
 */
uint_t* Engine::getTreeAncestor(){
	return devTreeAncestor;
}
/**
 * Array of past cumulative children
 */
uint_t* Engine::getTreeNumChildrenCumul(){
	return devTreeNumChildrenCumul;
}
/**
 * Array of the probability
 */
real_t* Engine::getTreeProb(){
	return devTreeProb;
}
/**
 * Array of the error in the demand
 */
real_t* Engine::getTreeErrorDemand(){
	return devTreeErrorDemand;
}
/**
 * Array of the error in the prices
 */
real_t* Engine::getTreeErrorPrices(){
	return devTreeErrorPrices;
}
/** ----GETTER'S OF NETWORK CONSTRAINTS----*/
/**
 * state/volume minimum
 */
real_t* Engine::getSysXmin(){
	return devSysXmin;
}
/**
 * state/volume maximum
 */
real_t* Engine::getSysXmax(){
	return devSysXmax;
}
/**
 * state/volume safe level
 */
real_t* Engine::getSysXs(){
	return devSysXs;
}
/**
 * dummy state/volume safe level
 */
real_t* Engine::getSysXsUpper(){
	return devSysXsUpper;
}
/**
 * actuator/control minimum
 */
real_t* Engine::getSysUmin(){
	return devSysUmin;
}
/**
 * actuator/control maximum
 */
real_t* Engine::getSysUmax(){
	return devSysUmax;
}
/**
 * cublasHandle
 */
hipblasHandle_t Engine::getCublasHandle(){
	return handle;
}

/**
 * status of price uncertainty
 */
bool Engine::getPriceUncertainty(){
	return priceUncertaintyFlag;
}
/**
 * status of the demand uncertanity
 */
bool Engine::getDemandUncertantiy(){
	return demandUncertaintyFlag;
}
/*  SETTER'S IN THE ENGINE  */
/*
 * Option for uncertainty in price
 * @param    priceUncertaintyFlag    true to include uncertainty (default)
 *                                   false to include uncertainty (default)
 */
void Engine::setPriceUncertaintyFlag(bool inputFlag){
	priceUncertaintyFlag = inputFlag;
}
/*
 * Option for uncertainty in demand
 * @param    demandUncertaintyFlag    true to include uncertainty (default)
 *                                    false to include uncertainty (default)
 */
void Engine::setDemandUncertaintyFlag(bool inputFlag){
	demandUncertaintyFlag = inputFlag;
}

void Engine::eliminateInputDistubanceCoupling(real_t* nominalDemand, real_t *nominalPrices){
	uint_t ns = ptrMyScenarioTree->getNumScenarios();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t nd = ptrMyNetwork->getNumDemands();
	uint_t N =  ptrMyScenarioTree->getPredHorizon();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t numNonleafNodes =  ptrMyScenarioTree->getNumNonleafNodes();
	real_t alpha = 1, beta = 0;
	real_t *devMatGd;
	real_t **devPtrMatGd, **devPtrVecE, **devPtrVecDemand;
	real_t **ptrMatGd = new real_t*[ns];
	real_t **ptrVecE = new real_t*[nodes];
	real_t **ptrVecDemand = new real_t*[nodes];
	real_t **ptrVecUhat = new real_t*[nodes];
	real_t *devVecDemand, *devVecDemandHat;
	real_t **devPtrVecUhat, *devVecDeltaUhat, *devVecZeta;
	real_t *devVecAlphaHat;
	real_t *devVecAlpha1;
	real_t *devVecAlpha;
	real_t *devVecAlphaBar;
	real_t *devMatRhat;
	uint_t *nodeStage = ptrMyScenarioTree->getStageNodes();
	uint_t *nodesPerStage = ptrMyScenarioTree->getNodesPerStage();
	uint_t *nodesPerStageCumul = ptrMyScenarioTree->getNodesPerStageCumul();
	uint_t iStageNodes, iStageCumulNodes, jNodes, iStage;


	_CUDA( hipMalloc((void**)&devVecDemand, nodes*nd*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDemandHat, N*nd*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatGd, ns*nx*nd*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devPtrMatGd, ns*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecE, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecDemand, nodes*sizeof(real_t*)));
	_CUDA( hipMalloc((void**)&devPtrVecUhat, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devVecAlphaHat, N*nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecAlpha1, nu*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecAlpha, nodes*nu*sizeof(real_t)));
	_CUDA( hipMalloc((void**)&devVecAlphaBar, nodes*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devMatRhat, nu*nv*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDeltaUhat, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecZeta, nu*nodes*sizeof(real_t)) );


	for (uint_t iScenario = 0; iScenario < ns; iScenario++){
		_CUDA( hipMemcpy(&devMatGd[iScenario*nx*nd], ptrMyNetwork->getMatGd(), nx*nd*sizeof(real_t), hipMemcpyHostToDevice) );
		ptrMatGd[iScenario] = &devMatGd[iScenario*nx*nd];
	}
	for( uint_t iNode = 0; iNode < nodes; iNode++){
		ptrVecE[iNode] = &devVecE[iNode*nx];
		ptrVecDemand[iNode] = &devVecDemand[iNode*nd];
		ptrVecUhat[iNode] = &devVecUhat[iNode*nu];
	}
	_CUDA( hipMemcpy(devPtrMatGd, ptrMatGd, ns*sizeof(real_t*), hipMemcpyHostToDevice));
	_CUDA( hipMemcpy(devPtrVecDemand, ptrVecDemand, nodes*sizeof(real_t*), hipMemcpyHostToDevice));
	_CUDA( hipMemcpy(devPtrVecE, ptrVecE, nodes*sizeof(real_t*), hipMemcpyHostToDevice));
	_CUDA( hipMemcpy(devPtrVecUhat, ptrVecUhat, nodes*sizeof(real_t*), hipMemcpyHostToDevice));
	// d(node) = dhat(stage) + d(node)
	// e = Gd*d
	_CUDA( hipMemcpy(devVecDemand, ptrMyScenarioTree->getErrorDemandArray(), nodes*nd*sizeof(real_t),
			hipMemcpyHostToDevice ));
	if(!demandUncertaintyFlag){
		_CUBLAS( hipblasSscal(handle, nu*nodes, &beta, devVecDemand, 1) );
	}
	_CUDA( hipMemcpy(devVecDemandHat, nominalDemand, N*nd*sizeof(real_t), hipMemcpyHostToDevice ));
	for (iStage = 0 ; iStage < N; iStage++){
		iStageCumulNodes = nodesPerStageCumul[iStage];
		iStageNodes = nodesPerStage[iStage];
		for(uint_t j = 0; j < iStageNodes; j++){
			jNodes = iStageCumulNodes + j;
			_CUBLAS( hipblasSaxpy(handle, nd, &alpha, &devVecDemandHat[iStage*nd], 1, &devVecDemand[jNodes*nd],1) );
		}
		_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nx, 1, nd, &alpha, (const real_t**)devPtrMatGd,
				nx, (const real_t**)&devPtrVecDemand[iStageCumulNodes], nd, &beta, &devPtrVecE[iStageCumulNodes],
				nx , iStageNodes));
	}
	// uhat = Lhat*d
	_CUBLAS(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nu, 1, nd, &alpha, (const real_t**)
			devPtrSysMatLhat, nu, (const real_t**)devPtrVecDemand, nd, &beta, devPtrVecUhat, nu , nodes));
	// alpha = alphaHat + alpha1 + errorprice
	_CUDA( hipMemcpy(devVecAlphaHat, nominalPrices, N*nu*sizeof(real_t), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devVecAlpha1, ptrMyNetwork->getAlpha(), nu*sizeof(real_t), hipMemcpyHostToDevice));
	_CUDA(hipMemcpy(devVecAlpha, ptrMyScenarioTree->getErrorPriceArray(), nu*nodes*sizeof(real_t), hipMemcpyHostToDevice));
	//_CUBLAS(hipblasSscal(handle,n,&al,d x,1));
	if(!priceUncertaintyFlag){
		_CUBLAS( hipblasSscal(handle, nu*nodes, &beta, devVecAlpha, 1) );
	}

	for(iStage = 0; iStage < N; iStage++){
		_CUBLAS( hipblasSaxpy(handle, nu, &alpha, devVecAlpha1, 1, &devVecAlphaHat[iStage*nu], 1) );
	}
	for(uint_t iNode = 0; iNode < nodes; iNode++){
		iStage = nodeStage[iNode];
		_CUBLAS( hipblasSaxpy(handle, nu, &alpha, &devVecAlphaHat[iStage*nu], 1 , &devVecAlpha[iNode*nu], 1));
	}
	//scaling with the weight
	real_t weightEconomical = ptrMySmpcConfig->getWeightEconomical();
	_CUBLAS( hipblasSscal(handle, nu*nodes, &weightEconomical, devVecAlpha, 1) );
	// alphaBar = L* (alpha)
	_CUBLAS( hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nv, nodes, nu, &alpha, (const real_t*) devSysMatL, nu,
			(const real_t*)devVecAlpha, nu, &beta, devVecAlphaBar, nv));
	_CUBLAS( hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nu, nv, nu, &alpha, (const real_t*) devSysCostW, nu,
			(const real_t*)devSysMatL, nu, &beta, devMatRhat, nu));
	// Beta
	calculateDiffUhat<<<nodes, nu>>>(devVecDeltaUhat, devVecUhat, devVecPreviousUhat, devTreeAncestor, nu, nodes);
	calculateZeta<<<nodes, nu>>>(devVecZeta, devVecDeltaUhat, devTreeProb, devTreeNumChildrenCumul, nu, numNonleafNodes, nodes);

	alpha = 2;
	_CUBLAS( hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nv, nodes, nu, &alpha, (const real_t *) devMatWv, nu,
			(const real_t *) devVecZeta, nu, &beta, devVecBeta, nv) );
	alpha = 1;

	for(uint_t iNode = 0; iNode < nodes; iNode++){
		real_t scale = ptrMyScenarioTree->getProbArray()[iNode];
		_CUBLAS( hipblasSaxpy(handle, nv, &scale, &devVecAlphaBar[nv*iNode], 1, &devVecBeta[nv*iNode],1) );
	}

	delete [] ptrMatGd;
	delete [] ptrVecE;
	delete [] ptrVecDemand;
	delete [] ptrVecUhat;
	_CUDA(hipFree(devVecDemand));
	_CUDA(hipFree(devVecDemandHat));
	_CUDA(hipFree(devMatGd));
	_CUDA(hipFree(devPtrMatGd));
	_CUDA(hipFree(devPtrVecE));
	_CUDA(hipFree(devPtrVecDemand));
	_CUDA(hipFree(devPtrVecUhat));
	_CUDA(hipFree(devVecAlpha));
	_CUDA(hipFree(devVecAlphaHat));
	_CUDA(hipFree(devVecAlpha1));
	_CUDA(hipFree(devVecAlphaBar));
	_CUDA(hipFree(devMatRhat));
	_CUDA(hipFree(devVecDeltaUhat));
	_CUDA(hipFree(devVecZeta));

	ptrMatGd = NULL;
	ptrVecE = NULL;
	ptrVecDemand = NULL;
	ptrVecUhat = NULL;
	devVecDemand = NULL;
	devVecDemandHat = NULL;
	devMatGd = NULL;
	devPtrMatGd = NULL;
	devPtrVecE = NULL;
	devPtrVecDemand = NULL;
	devPtrVecUhat = NULL;
	devVecAlphaHat = NULL;
	devVecAlpha = NULL;
	devVecAlpha1 = NULL;
	devVecAlphaBar = NULL;
	devMatRhat = NULL;
	devVecDeltaUhat = NULL;
	devVecZeta = NULL;
}

void Engine::updateStateControl(real_t* currentX, real_t* prevU, real_t* prevDemand){
	real_t alpha = 1;
	real_t beta = 0;
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nd = ptrMyNetwork->getNumDemands();
	uint_t nv = ptrMySmpcConfig->getNV();

	_CUDA( hipMemcpy(devVecCurrentState, currentX, ptrMyNetwork->getNumTanks()*sizeof(real_t),
			hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devVecPreviousControl, prevU, ptrMyNetwork->getNumControls()*sizeof(real_t),
			hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devVecPreviousDemand, prevDemand, ptrMyNetwork->getNumDemands()*sizeof(real_t),
			hipMemcpyHostToDevice));
	_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, nu, nd, &alpha, devSysMatLhat, nu, devVecPreviousDemand, 1,
			&beta, devVecPreviousUhat, 1) );
}

void Engine::inverseBatchMat(real_t** src, real_t** dst, uint_t n, uint_t batchSize){
	uint_t *P, *INFO;

	_CUDA(hipMalloc((void**)&P, n * batchSize * sizeof(uint_t)));
	_CUDA(hipMalloc((void**)&INFO, batchSize * sizeof(uint_t)));

	uint_t lda = n;

	real_t** x=(real_t**)malloc(batchSize*sizeof(real_t*));
	real_t* y=(real_t*)malloc(n*n*sizeof(real_t));


	_CUBLAS(hipblasSgetrfBatched(handle,n,src,lda,P,INFO,batchSize));

	uint_t INFOh[batchSize];

	_CUDA(hipMemcpy(INFOh,INFO,batchSize*sizeof(uint_t),hipMemcpyDeviceToHost));
	for (uint_t i = 0; i < batchSize; i++){
		if(INFOh[i] != 0)
		{
			fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
			hipDeviceReset();
			exit(EXIT_FAILURE);
		}
	}

	_CUBLAS(hipblasSgetriBatched(handle,n,(const real_t **)src,lda,P,dst,lda,INFO,batchSize));
	_CUDA(hipMemcpy(INFOh,INFO,batchSize*sizeof(uint_t),hipMemcpyDeviceToHost));

	for (uint_t i = 0; i < batchSize; i++)
		if(INFOh[i] != 0)
		{
			fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
			hipDeviceReset();
			exit(EXIT_FAILURE);
		}

	_CUDA(hipFree(P));
	_CUDA(hipFree(INFO));
	P = NULL;
	INFO = NULL;
}


void Engine::deallocateSystemDevice(){
	_CUDA( hipFree(devSysMatB) );
	_CUDA( hipFree(devSysMatL) );
	_CUDA( hipFree(devSysMatLhat) );
	_CUDA( hipFree(devSysMatF) );
	_CUDA( hipFree(devSysMatG) );
	_CUDA( hipFree(devSysXmin) );
	_CUDA( hipFree(devSysXmax) );
	_CUDA( hipFree(devSysXs) );
	_CUDA( hipFree(devSysXsUpper) );
	_CUDA( hipFree(devSysUmin) );
	_CUDA( hipFree(devSysUmax) );
	_CUDA( hipFree(devVecCurrentState));
	_CUDA( hipFree(devVecPreviousControl));
	_CUDA( hipFree(devVecPreviousUhat));
	_CUDA( hipFree(devVecPreviousDemand));
	_CUDA( hipFree(devMatWv) );

	_CUDA( hipFree(devPtrSysMatB) );
	_CUDA( hipFree(devPtrSysMatL) );
	_CUDA( hipFree(devPtrSysMatLhat) );
	_CUDA( hipFree(devPtrSysMatF) );
	_CUDA( hipFree(devPtrSysMatG) );

	devSysMatB = NULL;
	devSysMatL = NULL;
	devSysMatLhat = NULL;
	devSysMatF = NULL;
	devSysMatG = NULL;
	devSysXmin = NULL;
	devSysXmax = NULL;
	devSysXs = NULL;
	devSysXsUpper = NULL;
	devSysUmin = NULL;
	devSysUmax = NULL;
	devVecCurrentState = NULL;
	devVecPreviousControl = NULL;
	devVecPreviousUhat = NULL;
	devVecPreviousDemand = NULL;
	devMatWv = NULL;

	devPtrSysMatB = NULL;
	devPtrSysMatL = NULL;
	devPtrSysMatLhat = NULL;
	devPtrSysMatF = NULL;
	devPtrSysMatG = NULL;
}

void Engine::deallocateScenarioTreeDevice(){
	_CUDA( hipFree(devTreeStages) );
	_CUDA( hipFree(devTreeNodesPerStage));
	_CUDA( hipFree(devTreeLeaves) );
	_CUDA( hipFree(devTreeNodesPerStageCumul) );
	_CUDA( hipFree(devTreeNumChildren) );
	_CUDA( hipFree(devTreeNumChildrenCumul) );
	_CUDA( hipFree(devTreeErrorDemand) );
	_CUDA( hipFree(devTreeErrorPrices) );
	//_CUDA( hipFree(devForecastValue) );

	devTreeStages = NULL;
	devTreeNodesPerStage = NULL;
	devTreeLeaves = NULL;
	devTreeNodesPerStageCumul = NULL;
	devTreeNumChildren = NULL;
	devTreeNumChildrenCumul = NULL;
	devTreeErrorDemand = NULL;
	devTreeErrorPrices = NULL;
	//devForecastValue = NULL;
}
Engine::~Engine(){
	deallocateSystemDevice();
	deallocateScenarioTreeDevice();
	//delete ptrmyForecaster;
	//delete ptrMyNetwork;
	_CUDA(hipFree(devMatPhi));
	_CUDA(hipFree(devMatPsi));
	_CUDA(hipFree(devMatTheta));
	_CUDA(hipFree(devMatOmega));
	_CUDA(hipFree(devMatSigma));
	_CUDA(hipFree(devMatD));
	_CUDA(hipFree(devMatF));
	_CUDA(hipFree(devMatG));
	_CUDA(hipFree(devVecUhat));
	_CUDA(hipFree(devVecBeta));
	_CUDA(hipFree(devVecE));

	_CUDA(hipFree(devPtrMatPhi));
	_CUDA(hipFree(devPtrMatPsi));
	_CUDA(hipFree(devPtrMatTheta));
	_CUDA(hipFree(devPtrMatOmega));
	_CUDA(hipFree(devPtrMatSigma));
	_CUDA(hipFree(devPtrMatD));
	_CUDA(hipFree(devPtrMatF));
	_CUDA(hipFree(devPtrMatG));

	devMatPhi = NULL;
	devMatPsi = NULL;
	devMatTheta = NULL;
	devMatOmega = NULL;
	devMatSigma = NULL;
	devMatD = NULL;
	devMatF = NULL;
	devMatG = NULL;
	devVecUhat = NULL;
	devVecBeta = NULL;
	devVecE = NULL;

	devPtrMatPhi = NULL;
	devPtrMatPsi = NULL;
	devPtrMatTheta = NULL;
	devPtrMatOmega = NULL;
	devPtrMatSigma = NULL;
	devPtrMatD = NULL;
	devPtrMatF = NULL;
	devPtrMatG = NULL;
	//_CUBLAS(hipblasDestroy(handle));
	_CUBLAS(hipblasDestroy(handle));
}
