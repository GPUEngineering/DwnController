#include "hip/hip_runtime.h"
/*
 *    GPU-accelerated scenario-based stochastic MPC for the operational
 *    management of drinking water networks.
 *    Copyright (C) 2017 Ajay. K. Sampathirao and P. Sopasakis
 *
 *    This library is free software; you can redistribute it and/or
 *    modify it under the terms of the GNU Lesser General Public
 *    License as published by the Free Software Foundation; either
 *    version 2.1 of the License, or (at your option) any later version.
 *
 *    This library is distributed in the hope that it will be useful,
 *    but WITHOUT ANY WARRANTY; without even the implied warranty of
 *    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 *    Lesser General Public License for more details.
 *
 *    You should have received a copy of the GNU Lesser General Public
 *    License along with this library; if not, write to the Free Software
 *    Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA 02110-1301 USA
 */
#include <cuda_device_runtime_api.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "SmpcController.cuh"

SmpcController::SmpcController(Forecaster *myForecaster, Engine *myEngine, SmpcConfiguration *mySmpcConfig){
	ptrMyForecaster = myForecaster;
	ptrMyEngine = myEngine;
	ptrMySmpcConfig = mySmpcConfig;
	DwnNetwork* ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree* ptrMyScenarioTree = ptrMyEngine->getScenarioTree();

	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t ns = ptrMyScenarioTree->getNumScenarios();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	MAX_ITERATIONS  = 500;
	stepSize = 1e-4;

	_CUDA( hipMalloc((void**)&devVecX, nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecU, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecV, nv*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecAcceleratedXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecAcceleratedPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPrimalXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecPrimalPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDualXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDualPsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecUpdateXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecUpdatePsi, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devPrimalInfeasibilty, (2*nx + nu)*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecQ, ns*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecR, ns*nv*sizeof(real_t)) );

	_CUDA( hipMalloc((void**)&devPtrVecX, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecU, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecV, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecAcceleratedXi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecAcceleratedPsi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecPrimalXi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecPrimalPsi, nodes*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecQ, ns*sizeof(real_t*)) );
	_CUDA( hipMalloc((void**)&devPtrVecR, ns*sizeof(real_t*)) );

	real_t** ptrVecX = new real_t*[nodes];
	real_t** ptrVecU = new real_t*[nodes];
	real_t** ptrVecV = new real_t*[nodes];
	real_t** ptrVecAcceleratedXi = new real_t*[nodes];
	real_t** ptrVecAcceleratedPsi = new real_t*[nodes];
	real_t** ptrVecPrimalXi = new real_t*[nodes];
	real_t** ptrVecPrimalPsi = new real_t*[nodes];
	real_t** ptrVecQ = new real_t*[ns];
	real_t** ptrVecR = new real_t*[ns];

	for(int iLeaf = 0; iLeaf < ns; iLeaf++){
		ptrVecQ[iLeaf] = &devVecQ[iLeaf*nx];
		ptrVecR[iLeaf] = &devVecR[iLeaf*nv];
	}
	for(int iNode = 0; iNode < nodes; iNode++){
		ptrVecX[iNode] = &devVecX[iNode*nx];
		ptrVecU[iNode] = &devVecU[iNode*nu];
		ptrVecV[iNode] = &devVecV[iNode*nv];
		ptrVecAcceleratedXi[iNode] = &devVecAcceleratedXi[2*iNode*nx];
		ptrVecAcceleratedPsi[iNode] = &devVecAcceleratedPsi[iNode*nu];
		ptrVecPrimalXi[iNode] = &devVecPrimalXi[2*iNode*nx];
		ptrVecPrimalPsi[iNode] = &devVecPrimalPsi[iNode*nu];
	}

	_CUDA( hipMemset(devVecU, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecAcceleratedXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecAcceleratedPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecUpdateXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecUpdatePsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecDualXi, 0, 2*nx*nodes*sizeof(real_t)));
	_CUDA( hipMemset(devVecDualPsi, 0, nu*nodes*sizeof(real_t)) );

	_CUDA( hipMemcpy(devPtrVecX, ptrVecX, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecU, ptrVecU, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecV, ptrVecV, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecAcceleratedXi, ptrVecAcceleratedXi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecAcceleratedPsi, ptrVecAcceleratedPsi, nodes*sizeof(real_t*), hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecPrimalXi, ptrVecPrimalXi, nodes*sizeof(real_t*),hipMemcpyHostToDevice) );
	_CUDA( hipMemcpy(devPtrVecPrimalPsi, ptrVecPrimalPsi, nodes*sizeof(real_t*),hipMemcpyHostToDevice) );

	delete [] ptrVecX;
	delete [] ptrVecU;
	delete [] ptrVecV;
	delete [] ptrVecAcceleratedXi;
	delete [] ptrVecAcceleratedPsi;
	delete [] ptrVecPrimalXi;
	delete [] ptrVecPrimalPsi;
	ptrVecX = NULL;
	ptrVecU = NULL;
	ptrVecV = NULL;
	ptrVecAcceleratedXi = NULL;
	ptrVecAcceleratedPsi = NULL;
	ptrVecPrimalXi = NULL;
	ptrVecPrimalPsi = NULL;
	ptrMyNetwork = NULL;
	ptrMyScenarioTree = NULL;
}

void SmpcController::dualExtrapolationStep(real_t lambda){
	DwnNetwork* ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree* ptrMyScenarioTree = ptrMyEngine->getScenarioTree();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	real_t alpha;
	// w = (1 + \lambda)y_k - \lambda y_{k-1}
	_CUDA(hipMemcpy(devVecAcceleratedXi, devVecUpdateXi, 2*nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice) );
	_CUDA(hipMemcpy(devVecAcceleratedPsi, devVecUpdatePsi, nu*nodes*sizeof(real_t), hipMemcpyDeviceToDevice) );
	alpha = 1 + lambda;
	_CUBLAS(hipblasSscal(ptrMyEngine->handle, 2*nx*nodes, &alpha, devVecAcceleratedXi, 1));
	_CUBLAS(hipblasSscal(ptrMyEngine->handle, nu*nodes, &alpha, devVecAcceleratedPsi, 1));
	alpha = -lambda;
	_CUBLAS(hipblasSaxpy(ptrMyEngine->handle, 2*nx*nodes, &alpha, devVecXi, 1, devVecAcceleratedXi, 1));
	_CUBLAS(hipblasSaxpy(ptrMyEngine->handle, nu*nodes, &alpha, devVecPsi, 1, devVecAcceleratedPsi, 1));
	// y_{k} = y_{k-1}
	_CUDA(hipMemcpy(devVecXi, devVecUpdateXi, 2*nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	_CUDA(hipMemcpy(devVecPsi, devVecUpdatePsi, nu*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
}

void SmpcController::solveStep(){
	DwnNetwork *ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree *ptrMyScenarioTree = ptrMyEngine->getScenarioTree();
	real_t *devTempVecR, *devTempVecQ;
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	uint_t nv = ptrMySmpcConfig->getNV();
	uint_t ns = ptrMyScenarioTree->getNumScenarios();
	uint_t N =  ptrMyScenarioTree->getPredHorizon();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t *nodesPerStage = ptrMyScenarioTree->getNodesPerStage();
	uint_t *nodesPerStageCumul = ptrMyScenarioTree->getNodesPerStageCumul();
	uint_t iStageCumulNodes, iStageNodes, prevStageNodes, prevStageCumulNodes;
	real_t scale[2] = {-0.5, 1};
	real_t alpha = 1;
	real_t beta = 0;

	cout<< nx << " " << nu << " " << ns << endl;
	_CUDA( hipMalloc((void**)&devTempVecQ, ns*nx*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devTempVecR, ns*nv*sizeof(real_t)) );
	_CUDA( hipMemcpy(ptrMyEngine->getMatSigma(), ptrMyEngine->getVecBeta(), nv*nodes*sizeof(real_t),
			hipMemcpyDeviceToDevice) );

	for(int iStage = N-1;iStage > -1;iStage--){
		iStageCumulNodes = nodesPerStageCumul[iStage];
		iStageNodes = nodesPerStage[iStage];
		if(iStage < N-1){
			// sigma=sigma+r
			_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), iStageNodes*nv, &alpha, devVecR, 1,
					&ptrMyEngine->getMatSigma()[iStageCumulNodes*nv],1));
		}
		// v=Omega*sigma
		_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nv,
				&scale[0], (const float**)&ptrMyEngine->getPtrMatOmega()[iStageCumulNodes], nv,
				(const float**)&ptrMyEngine->getPtrMatSigma()[iStageCumulNodes], nv, &beta,
				&devPtrVecV[iStageCumulNodes], nv, iStageNodes));

		if(iStage < N-1){
			// v=Theta*q+v
			_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nx,
					&alpha, (const float**)&ptrMyEngine->getPtrMatTheta()[iStageCumulNodes], nv,
					(const float**)devPtrVecQ, nx, &alpha, &devPtrVecV[iStageCumulNodes], nv, iStageNodes));
		}

		// v=Psi*psi+v
		_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nu, &alpha,
				(const float**)&ptrMyEngine->getPtrMatPsi()[iStageCumulNodes], nv,
				(const float**)&devPtrVecAcceleratedPsi[iStageCumulNodes], nu, &alpha, &devPtrVecV
				[iStageCumulNodes], nv, iStageNodes));

		// v=Phi*xi+v
		_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, 2*nx, &alpha,
				(const float**)&ptrMyEngine->getPtrMatPhi()[iStageCumulNodes], nv, (const float**)
				&devPtrVecAcceleratedXi[iStageCumulNodes], 2*nx, &alpha, &devPtrVecV[iStageCumulNodes],
				nv, iStageNodes));

		// r=sigma
		_CUDA(hipMemcpy(devVecR, &ptrMyEngine->getMatSigma()[iStageCumulNodes*nv], nv*iStageNodes*sizeof(real_t),
				hipMemcpyDeviceToDevice));

		// r=D*xi+r
		_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, 2*nx, &alpha,
				(const float**)&ptrMyEngine->getPtrMatD()[iStageCumulNodes], nv, (const float**)
				&devPtrVecAcceleratedXi[iStageCumulNodes], 2*nx, &alpha, devPtrVecR, nv, iStageNodes));

		// r=f*psi+r
		_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nu, &alpha,
				(const float**)&ptrMyEngine->getPtrMatF()[iStageCumulNodes], nv, (const float**)
				&devPtrVecAcceleratedPsi[iStageCumulNodes], nu, &alpha, devPtrVecR, nv, iStageNodes));

		if(iStage < N-1){
			// r=g*q+r
			_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nv, 1, nx, &alpha,
					(const float**)&ptrMyEngine->getPtrMatG()[iStageCumulNodes], nv, (const float**)devPtrVecQ,
					nx, &alpha, devPtrVecR, nv, iStageNodes));
		}

		if(iStage < N-1){
			// q=F'xi+q
			_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_T, HIPBLAS_OP_N, nx, 1, 2*nx, &alpha,
					(const float**)&ptrMyEngine->getPtrMatF()[iStageCumulNodes], 2*nx, (const float**)
					&devPtrVecAcceleratedXi[iStageCumulNodes], 2*nx, &alpha, devPtrVecQ, nx, iStageNodes));
		}else{
			// q=F'xi
			_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_T, HIPBLAS_OP_N, nx, 1, 2*nx, &alpha,
					(const float**)&ptrMyEngine->getPtrMatF()[iStageCumulNodes], 2*nx, (const float**)
					&devPtrVecAcceleratedXi[iStageCumulNodes], 2*nx, &beta, devPtrVecQ, nx, iStageNodes));
		}
		if(iStage > 0){
			prevStageNodes = nodesPerStage[iStage - 1];
			prevStageCumulNodes = nodesPerStageCumul[iStage - 1];
			if( (iStageNodes - prevStageNodes) > 0 ){
				solveSumChildren<<<prevStageNodes, nx>>>(devVecQ, devTempVecQ, ptrMyEngine->getTreeNumChildren(),
						ptrMyEngine->getTreeNumChildrenCumul(), prevStageCumulNodes, prevStageNodes, iStage - 1, nx);
				solveSumChildren<<<prevStageNodes, nx>>>(devVecR, devTempVecR, ptrMyEngine->getTreeNumChildren(),
						ptrMyEngine->getTreeNumChildrenCumul(), prevStageCumulNodes, prevStageNodes, iStage - 1 , nv);
				_CUDA(hipMemcpy(devVecR, devTempVecR, prevStageNodes*nv*sizeof(real_t), hipMemcpyDeviceToDevice));
				_CUDA(hipMemcpy(devVecQ, devTempVecQ, prevStageNodes*nx*sizeof(real_t), hipMemcpyDeviceToDevice));
			}
		}
	}

	// Forward substitution
	_CUDA(hipMemcpy(devVecU, ptrMyEngine->getVecUhat(), nodes*nu*sizeof(real_t), hipMemcpyDeviceToDevice));

	for(int iStage = 0;iStage < N;iStage++){
		iStageNodes = nodesPerStage[iStage];
		iStageCumulNodes = nodesPerStageCumul[iStage];
		if(iStage == 0){
			// x=p, u=h
			_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nv, &alpha, ptrMyEngine->getVecPreviousUhat(),
					1, devVecV, 1));
			_CUDA( hipMemcpy(devVecX, ptrMyEngine->devVecCurrentState, nx*sizeof(real_t), hipMemcpyDeviceToDevice) );
			// x=x+w
			_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nx, &alpha, ptrMyEngine->getVecE(), 1, devVecX, 1));
			// u=Lv+\hat{u}
			_CUBLAS(hipblasSgemv(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, nu, nv, &alpha,
					ptrMyEngine->getSysMatL(), nu, devVecV, 1, &alpha, devVecU, 1) );
			// x=x+Bu
			_CUBLAS(hipblasSgemv(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, nx, nu, &alpha,
					ptrMyEngine->getSysMatB(), nx, devVecU, 1, &alpha, devVecX, 1) );
		}else{
			prevStageCumulNodes = nodesPerStageCumul[iStage - 1];
			if((nodesPerStage[iStage] - nodesPerStage[iStage-1]) > 0){
				// v_k=v_{k-1}+v_k
				solveChildNodesUpdate<<<iStageNodes, nv>>>(&devVecV[prevStageCumulNodes*nv], &devVecV[iStageCumulNodes*nv],
						ptrMyScenarioTree->getAncestorArray(), iStageCumulNodes, nv);
				// u_k=Lv_k+\hat{u}_k
				_CUBLAS(hipblasSgemm(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nu, iStageNodes, nv,
						&alpha, ptrMyEngine->getSysMatL(), nu, &devVecV[iStageCumulNodes*nv], nv, &alpha,
						&devVecU[iStageCumulNodes*nu], nu));
				// x=w
				_CUDA(hipMemcpy(&devVecX[iStageCumulNodes*nx], &ptrMyEngine->devVecE[iStageCumulNodes*nx],
						iStageNodes*nx*sizeof(real_t), hipMemcpyDeviceToDevice));
				// x=x+Bu
				_CUBLAS(hipblasSgemm(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nx, iStageCumulNodes, nu, &alpha,
						ptrMyEngine->getSysMatB(), nx, &devVecU[iStageCumulNodes*nu], nu, &alpha, &devVecX[iStageCumulNodes*nx], nx));
				// x_{k+1}=x_k
				solveChildNodesUpdate<<<iStageNodes, nx>>>(&devVecX[prevStageCumulNodes*nx], &devVecX[iStageCumulNodes*nx],
						ptrMyScenarioTree->getAncestorArray(), iStageCumulNodes, nx);
			}else{
				// v_k=v_{k-1}+v_k
				_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nv*iStageNodes, &alpha, &devVecV[prevStageCumulNodes*nv], 1,
						&devVecV[iStageCumulNodes*nv], 1));
				// u_k=Lv_k+\hat{u}_k
				_CUBLAS(hipblasSgemm(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nu, iStageNodes, nv, &alpha,
						ptrMyEngine->getSysMatL(), nu, &devVecV[iStageCumulNodes*nv], nv, &alpha, &devVecU[iStageCumulNodes*nu], nu));
				// x_{k+1}=x_{k}
				_CUDA(hipMemcpy(&devVecX[iStageCumulNodes*nx], &devVecX[prevStageCumulNodes*nx], nx*iStageNodes*sizeof(real_t),
						hipMemcpyDeviceToDevice));
				// x=x+w
				_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nx*iStageNodes, &alpha, &ptrMyEngine->getVecE()
						[iStageCumulNodes*nx], 1, &devVecX[iStageCumulNodes*nx], 1));
				// x=x+Bu
				_CUBLAS(hipblasSgemm(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nx, iStageNodes, nu, &alpha,
						ptrMyEngine->getSysMatB(), nx, &devVecU[iStageCumulNodes*nu], nu, &alpha, &devVecX[iStageCumulNodes*nx], nx));
			}
		}
	}/**/

	_CUDA(hipFree(devTempVecQ));
	_CUDA(hipFree(devTempVecR));
	devTempVecQ = NULL;
	ptrMyNetwork = NULL;
	ptrMyScenarioTree = NULL;
	ptrMyNetwork = NULL;

	//free(ptr_x_c);
	//free(x_c);
	//free(y_c);
}

void SmpcController::proximalFunG(){
	DwnNetwork *ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree *ptrMyScenarioTree = ptrMyEngine->getScenarioTree();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	real_t alpha = 1;
	real_t negAlpha = -1;
	real_t beta = 0;
	real_t penaltyScalar;
	real_t invLambda = 1/stepSize;
	real_t distanceXs, distanceXcst;
	real_t *devSuffleVecXi;
	real_t *devVecDiffXi;
	_CUDA( hipMalloc((void**)&devSuffleVecXi, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMalloc((void**)&devVecDiffXi, 2*nx*nodes*sizeof(real_t)) );
	// primalDual = Hx
	_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 2*nx, 1, nx, &alpha, (const float**)
			ptrMyEngine->getPtrSysMatF(), 2*nx, (const float**)devPtrVecX, 2*nx, &beta, devPtrVecPrimalXi, 2*nx, nodes) );
	_CUBLAS(hipblasSgemmBatched(ptrMyEngine->getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, nu, 1, nu, &alpha, (const float**)
			ptrMyEngine->getPtrSysMatG(), nu, (const float**)devPtrVecU, nu, &beta, devPtrVecPrimalPsi, nu, nodes) );
	// Hx + \lambda^{-1}w
	_CUDA( hipMemcpy(devVecDualXi, devPtrVecPrimalXi, 2*nodes*nx*sizeof(real_t), hipMemcpyDeviceToDevice) );
	_CUDA( hipMemcpy(devVecDualPsi, devPtrVecPrimalPsi, nodes*nu*sizeof(real_t), hipMemcpyDeviceToDevice) );
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), 2*nx*nodes, &invLambda, devVecAcceleratedXi, 1, devVecDualXi, 1) );
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu*nodes, &invLambda, devVecAcceleratedPsi, 1, devVecDualPsi, 1) );

	_CUDA( hipMemcpy(devVecDiffXi, devVecDualXi, 2*nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice) );
	// proj(xi|X), proj(xi|Xsafe)
	projectionBox<<<nodes, nx>>>(devVecDualXi, ptrMyEngine->getSysXmin(), ptrMyEngine->getSysXmax(), 2*nx, 0, nx*nodes);
	projectionBox<<<nodes, nx>>>(devVecDualXi, ptrMyEngine->getSysXs(), ptrMyEngine->getSysXsUpper(), 2*nx, nx, nx*nodes);
	// x-proj(x)
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), 2*nx*nodes, &negAlpha, devVecDualXi, 1, devVecDiffXi, 1) );
	shuffleVector<<<nodes, 2*nx>>>(devSuffleVecXi, devVecDiffXi, nx, 2, nodes);
	_CUBLAS(hipblasSnrm2(ptrMyEngine->getCublasHandle(), nx*nodes, devSuffleVecXi, 1, &distanceXcst));
	if(distanceXcst > invLambda*ptrMySmpcConfig->getPenaltyState()){
		penaltyScalar = 1-1/distanceXcst;
		additionVectorOffset<<<nodes, nx>>>(devVecDualXi, devVecDiffXi, penaltyScalar, 2*nx, 0, nx*nodes);
	}
	_CUBLAS(hipblasSnrm2(ptrMyEngine->getCublasHandle(), nx*nodes, &devSuffleVecXi[nx*nodes], 1, &distanceXs));
	if(distanceXs > invLambda*ptrMySmpcConfig->getPenaltySafety()){
		penaltyScalar = 1-1/distanceXs;
		additionVectorOffset<<<nodes, nx>>>(devVecDualXi, devVecDiffXi, penaltyScalar, 2*nx, nx, nx*nodes);
	}
	projectionBox<<<nodes, nu>>>(devVecDualPsi, ptrMyEngine->getSysUmin(), ptrMyEngine->getSysUmax(), nu, 0, nu*nodes);
	_CUDA( hipFree(devSuffleVecXi) );
	_CUDA( hipFree(devVecDiffXi) );
	devSuffleVecXi = NULL;
	devVecDiffXi = NULL;
	ptrMyNetwork = NULL;
	ptrMyScenarioTree = NULL;
}

void SmpcController::dualUpdate(){
	DwnNetwork *ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree *ptrMyScenarioTree = ptrMyEngine->getScenarioTree();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	real_t negAlpha = -1;
	//Hx - z
	_CUDA(hipMemcpy(devPrimalInfeasibilty, devVecPrimalXi, 2*nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	_CUDA(hipMemcpy(&devPrimalInfeasibilty[2*nx*nodes], devVecPrimalPsi, nu*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), 2*nx*nodes, &negAlpha, devVecDualXi, 1, devPrimalInfeasibilty, 1));
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu*nodes, &negAlpha, devVecDualPsi, 1, &devPrimalInfeasibilty[2*nx*nodes], 1));
	// y = w + \lambda(Hx - z)
	_CUDA( hipMemcpy(devVecUpdateXi, devVecAcceleratedXi, 2*nx*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	_CUDA( hipMemcpy(devVecUpdatePsi, devVecAcceleratedPsi, nu*nodes*sizeof(real_t), hipMemcpyDeviceToDevice));
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), 2*nx*nodes, &stepSize, devPrimalInfeasibilty, 1, devVecUpdateXi, 1) );
	_CUBLAS(hipblasSaxpy(ptrMyEngine->getCublasHandle(), nu*nodes, &stepSize, &devPrimalInfeasibilty[2*nx*nodes], 1,
			devVecUpdatePsi, 1) );
}


void SmpcController::algorithmApg(){
	DwnNetwork *ptrMyNetwork = ptrMyEngine->getDwnNetwork();
	ScenarioTree *ptrMyScenarioTree = ptrMyEngine->getScenarioTree();
	uint_t nodes = ptrMyScenarioTree->getNumNodes();
	uint_t nx = ptrMyNetwork->getNumTanks();
	uint_t nu = ptrMyNetwork->getNumControls();
	_CUDA( hipMemset(devVecXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecAcceleratedXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecAcceleratedPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecUpdateXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecUpdatePsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalXi, 0, 2*nx*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecPrimalPsi, 0, nu*nodes*sizeof(real_t)) );
	_CUDA( hipMemset(devVecDualXi, 0, 2*nx*nodes*sizeof(real_t)));
	_CUDA( hipMemset(devVecDualPsi, 0, nu*nodes*sizeof(real_t)) );

	real_t theta[2] = {1, 1};
	real_t lambda;

	for (int iter = 0; iter < MAX_ITERATIONS; iter++){
		lambda = theta[1]*(1/theta[0] - 1);
		dualExtrapolationStep(lambda);
		solveStep();
		proximalFunG();
		dualUpdate();
		theta[1] = 0.5*(sqrt(pow(theta[0], 4) + 4*theta[0]) - pow(theta[0], 2));
	}

	//dualExtrapolationStep(devPtrVecAcceleratedXi, devVecXi, devVecUpdateXi, lambda, 2*nx*nodes);
	//dualExtrapolationStep(devPtrVecAcceleratedPsi, devVecPsi, devVecUpdatePsi, lambda, nu*nodes);
	//dualUpdate(devVecUpdateXi, devPtrVecAcceleratedXi, devPtrVecPrimalXi, devVecDualXi, stepSize, 2*nodes*nx);
	//dualUpdate(devVecUpdatePsi, devPtrVecAcceleratedPsi, devPtrVecPrimalPsi, devVecDualPsi, stepSize, nodes*nu);
}

void SmpcController::controllerSmpc(){
	ptrMyEngine->updateStateControl(ptrMySmpcConfig->getCurrentX(), ptrMySmpcConfig->getPrevU(),
			ptrMySmpcConfig->getPrevUhat());
	ptrMyEngine->eliminateInputDistubanceCoupling(ptrMyForecaster->getNominalDemand(),
			ptrMyForecaster->getNominalPrices());
	algorithmApg();
}

SmpcController::~SmpcController(){
	_CUDA( hipFree(devVecX) );
	_CUDA( hipFree(devVecU) );
	_CUDA( hipFree(devVecV) );
	_CUDA( hipFree(devVecXi) );
	_CUDA( hipFree(devVecPsi) );
	_CUDA( hipFree(devVecAcceleratedXi) );
	_CUDA( hipFree(devVecAcceleratedPsi) );
	_CUDA( hipFree(devVecPrimalXi) );
	_CUDA( hipFree(devVecPrimalPsi) );
	_CUDA( hipFree(devVecDualXi) );
	_CUDA( hipFree(devVecDualPsi) );
	_CUDA( hipFree(devVecUpdateXi) );
	_CUDA( hipFree(devVecUpdatePsi) );
	_CUDA( hipFree(devPrimalInfeasibilty) );

	_CUDA( hipFree(devPtrVecX) );
	_CUDA( hipFree(devPtrVecU) );
	_CUDA( hipFree(devPtrVecV) );
	_CUDA( hipFree(devPtrVecAcceleratedXi) );
	_CUDA( hipFree(devPtrVecAcceleratedPsi) );
	_CUDA( hipFree(devPtrVecPrimalXi) );
	_CUDA( hipFree(devPtrVecPrimalPsi) );
}
